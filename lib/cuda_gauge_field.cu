#include "hip/hip_runtime.h"
#include <string.h>
#include <gauge_field.h>
#include <typeinfo>
#include <blas_quda.h>

namespace quda {

  cudaGaugeField::cudaGaugeField(const GaugeFieldParam &param) :
    GaugeField(param), gauge(0), even(0), odd(0)
  {
    if ((order == QUDA_QDP_GAUGE_ORDER || order == QUDA_QDPJIT_GAUGE_ORDER) &&
        create != QUDA_REFERENCE_FIELD_CREATE) {
      errorQuda("QDP ordering only supported for reference fields");
    }

    if (order == QUDA_QDP_GAUGE_ORDER ||
	order == QUDA_TIFR_GAUGE_ORDER || order == QUDA_TIFR_PADDED_GAUGE_ORDER ||
	order == QUDA_BQCD_GAUGE_ORDER || order == QUDA_CPS_WILSON_GAUGE_ORDER)
      errorQuda("Field ordering %d presently disabled for this type", order);

#ifdef MULTI_GPU
    if (link_type != QUDA_ASQTAD_MOM_LINKS &&
	ghostExchange == QUDA_GHOST_EXCHANGE_PAD &&
	isNative()) {
      bool pad_check = true;
      for (int i=0; i<nDim; i++) {
	// when we have coarse links we need to double the pad since we're storing forwards and backwards links
	int minimum_pad = nFace*surfaceCB[i] * (geometry == QUDA_COARSE_GEOMETRY ? 2 : 1);
	if (pad < minimum_pad) pad_check = false;
	if (!pad_check)
	  errorQuda("cudaGaugeField being constructed with insufficient padding (%d < %d)\n", pad, minimum_pad);
      }
    }
#endif

    if(create != QUDA_NULL_FIELD_CREATE &&
        create != QUDA_ZERO_FIELD_CREATE &&
        create != QUDA_REFERENCE_FIELD_CREATE){
      errorQuda("ERROR: create type(%d) not supported yet\n", create);
    }

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      gauge = pool_device_malloc(bytes);
      if (create == QUDA_ZERO_FIELD_CREATE) hipMemset(gauge, 0, bytes);
    } else {
      gauge = param.gauge;
    }

    if ( !isNative() ) {
      for (int i=0; i<nDim; i++) {
        size_t nbytes = nFace * surface[i] * nInternal * precision;
        ghost[i] = nbytes ? pool_device_malloc(nbytes) : nullptr;
	ghost[i+4] = (nbytes && geometry == QUDA_COARSE_GEOMETRY) ? pool_device_malloc(nbytes) : nullptr;
      }
    }

    if (ghostExchange == QUDA_GHOST_EXCHANGE_PAD) {
      if (create == QUDA_REFERENCE_FIELD_CREATE) exchangeGhost(geometry == QUDA_VECTOR_GEOMETRY ? QUDA_LINK_BACKWARDS : QUDA_LINK_BIDIRECTIONAL);
    }

    even = gauge;
    odd = (char*)gauge + bytes/2; 

#ifdef USE_TEXTURE_OBJECTS
    createTexObject(tex, gauge, true);
    createTexObject(evenTex, even, false);
    createTexObject(oddTex, odd, false);
    if(reconstruct == QUDA_RECONSTRUCT_13 || reconstruct == QUDA_RECONSTRUCT_9)
    {  // Create texture objects for the phases
      bool isPhase = true;
      createTexObject(phaseTex, (char*)gauge + phase_offset, true, isPhase);
      createTexObject(evenPhaseTex, (char*)even + phase_offset, false, isPhase);
      createTexObject(oddPhaseTex, (char*)odd + phase_offset, false, isPhase);
    }
#endif

  }

#ifdef USE_TEXTURE_OBJECTS
  void cudaGaugeField::createTexObject(hipTextureObject_t &tex, void *field, bool full, bool isPhase) {

    if( isNative() ){
      // create the texture for the field components
      hipChannelFormatDesc desc;
      memset(&desc, 0, sizeof(hipChannelFormatDesc));
      if (precision == QUDA_SINGLE_PRECISION) desc.f = hipChannelFormatKindFloat;
      else desc.f = hipChannelFormatKindSigned; // half is short, double is int2

      int texel_size = 1;
      if (isPhase) {
        if (precision == QUDA_DOUBLE_PRECISION) {
          desc.x = 8*sizeof(int);
          desc.y = 8*sizeof(int);
          desc.z = 0;
          desc.w = 0;
          texel_size = 2*sizeof(int);
        } else {
          desc.x = 8*precision;
          desc.y = desc.z = desc.w = 0;
          texel_size = precision;
        }
      } else {
        // always four components regardless of precision
        if (precision == QUDA_DOUBLE_PRECISION) {
          desc.x = 8*sizeof(int);
          desc.y = 8*sizeof(int);
          desc.z = 8*sizeof(int);
          desc.w = 8*sizeof(int);
	  texel_size = 4*sizeof(int);
        } else {
          desc.x = 8*precision;
          desc.y = 8*precision;
          desc.z = (reconstruct == 18 || reconstruct == 10) ? 0 : 8*precision; // float2 or short2 for 18 reconstruct
          desc.w = (reconstruct == 18 || reconstruct == 10) ? 0 : 8*precision;
          texel_size = (reconstruct == 18 || reconstruct == 10 ? 2 : 4) * precision;
        }
      }

      hipResourceDesc resDesc;
      memset(&resDesc, 0, sizeof(resDesc));
      resDesc.resType = hipResourceTypeLinear;
      resDesc.res.linear.devPtr = field;
      resDesc.res.linear.desc = desc;
      resDesc.res.linear.sizeInBytes = isPhase ? phase_bytes/(!full ? 2 : 1) : (bytes-phase_bytes)/(!full ? 2 : 1);

      unsigned long texels = resDesc.res.linear.sizeInBytes / texel_size;
      if (texels > (unsigned)deviceProp.maxTexture1DLinear) {
	errorQuda("Attempting to bind too large a texture %lu > %d", texels, deviceProp.maxTexture1DLinear);
      }

      hipTextureDesc texDesc;
      memset(&texDesc, 0, sizeof(texDesc));
      if (precision == QUDA_HALF_PRECISION) texDesc.readMode = hipReadModeNormalizedFloat;
      else texDesc.readMode = hipReadModeElementType;

      hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
      checkCudaError();
    }
  }

  void cudaGaugeField::destroyTexObject() {
    if( isNative() ){
      hipDestroyTextureObject(evenTex);
      hipDestroyTextureObject(oddTex);
      if(reconstruct == QUDA_RECONSTRUCT_9 || reconstruct == QUDA_RECONSTRUCT_13){
        hipDestroyTextureObject(evenPhaseTex);
        hipDestroyTextureObject(oddPhaseTex);
      }
      checkCudaError();
    }
  }
#endif

  cudaGaugeField::~cudaGaugeField()
  {
#ifdef USE_TEXTURE_OBJECTS
    destroyTexObject();
#endif

    destroyComms();

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      if (gauge) pool_device_free(gauge);
    }

    if ( !isNative() ) {
      for (int i=0; i<nDim; i++) {
        if (ghost[i]) pool_device_free(ghost[i]);
        if (ghost[i+4] && geometry == QUDA_COARSE_GEOMETRY) pool_device_free(ghost[i]);
      }
    }

  }

  // This does the exchange of the gauge field ghost zone and places it
  // into the ghost array.
  void cudaGaugeField::exchangeGhost(QudaLinkDirection link_direction) {
    if (ghostExchange != QUDA_GHOST_EXCHANGE_PAD)
      errorQuda("Cannot call exchangeGhost with ghostExchange=%d",
		ghostExchange);

    if (geometry != QUDA_VECTOR_GEOMETRY && geometry != QUDA_COARSE_GEOMETRY)
      errorQuda("Cannot exchange for %d geometry gauge field", geometry);

    if ( (link_direction == QUDA_LINK_BIDIRECTIONAL || link_direction == QUDA_LINK_FORWARDS) && geometry != QUDA_COARSE_GEOMETRY)
      errorQuda("Cannot request exchange of forward links on non-coarse geometry");

    void *ghost_[2*QUDA_MAX_DIM];
    void *send[2*QUDA_MAX_DIM];
    for (int d=0; d<nDim; d++) {
      ghost_[d] = isNative() ? pool_device_malloc(nFace*surface[d]*nInternal*precision) : ghost[d];
      send[d] = pool_device_malloc(nFace*surface[d]*nInternal*precision);
      if (geometry == QUDA_COARSE_GEOMETRY) { // bi-directional links
	ghost_[d+4] = isNative() ? pool_device_malloc(nFace*surface[d]*nInternal*precision) : ghost[d+4];
	send[d+4] = pool_device_malloc(nFace*surface[d]*nInternal*precision);
      }
    }

    if (link_direction == QUDA_LINK_BACKWARDS || link_direction == QUDA_LINK_BIDIRECTIONAL) {
      // get the links into contiguous buffers
      extractGaugeGhost(*this, send, true);

      // communicate between nodes
      exchange(ghost_, send, QUDA_FORWARDS);
    }

    // repeat if requested and links are bi-directional
    if (link_direction == QUDA_LINK_FORWARDS || link_direction == QUDA_LINK_BIDIRECTIONAL) {
      extractGaugeGhost(*this, send, true, nDim);
      exchange(ghost_+nDim, send+nDim, QUDA_FORWARDS);
    }

    for (int d=0; d<geometry; d++) pool_device_free(send[d]);

    if (isNative()) {
      // copy from ghost into the padded region in gauge
      if (link_direction == QUDA_LINK_BACKWARDS || link_direction == QUDA_LINK_BIDIRECTIONAL) copyGenericGauge(*this, *this, QUDA_CUDA_FIELD_LOCATION, 0, 0, 0, ghost_, 1);

      // repeat for the second set if bi-directional
      if (link_direction == QUDA_LINK_FORWARDS || link_direction == QUDA_LINK_BIDIRECTIONAL) copyGenericGauge(*this, *this, QUDA_CUDA_FIELD_LOCATION, 0, 0, 0, ghost_, 3);
    }

    if (isNative()) for (int d=0; d<geometry; d++) pool_device_free(ghost_[d]);
  }

  // This does the opposite of exchangeGhost and sends back the ghost
  // zone to the node from which it came and injects it back into the
  // field
  void cudaGaugeField::injectGhost(QudaLinkDirection link_direction) {
    if (ghostExchange != QUDA_GHOST_EXCHANGE_PAD)
      errorQuda("Cannot call exchangeGhost with ghostExchange=%d",
		ghostExchange);

    if (geometry != QUDA_VECTOR_GEOMETRY && geometry != QUDA_COARSE_GEOMETRY)
      errorQuda("Cannot exchange for %d geometry gauge field", geometry);

    if (link_direction != QUDA_LINK_BACKWARDS)
      errorQuda("link_direction = %d not supported", link_direction);

    void *ghost_[QUDA_MAX_DIM];
    void *recv[QUDA_MAX_DIM];
    for (int d=0; d<nDim; d++) {
      ghost_[d] = isNative() ? pool_device_malloc(nFace*surface[d]*nInternal*precision) : ghost[d];
      recv[d] = pool_device_malloc(nFace*surface[d]*nInternal*precision);
    }

    if (isNative()) {
      // copy from padded region in gauge field into ghost
      copyGenericGauge(*this, *this, QUDA_CUDA_FIELD_LOCATION, 0, 0, ghost_, 0, 1);
    }

    // communicate between nodes
    exchange(recv, ghost_, QUDA_BACKWARDS);

    // get the links into contiguous buffers
    extractGaugeGhost(*this, recv, false);

    for (int d=0; d<nDim; d++) {
      pool_device_free(recv[d]);
      if (isNative()) pool_device_free(ghost_[d]);
    }
  }

  void cudaGaugeField::allocateGhostBuffer(const int *R, bool no_comms_fill) const
  {
    createGhostZone(R, no_comms_fill);
    LatticeField::allocateGhostBuffer(ghost_bytes);
  }

  void cudaGaugeField::createComms(const int *R, bool no_comms_fill)
  {
    allocateGhostBuffer(R, no_comms_fill); // allocate the ghost buffer if not yet allocated

    // ascertain if this instance needs it comms buffers to be updated
    bool comms_reset = ghost_field_reset || // FIXME add send buffer check
      (my_face_h[0] != ghost_pinned_buffer_h[0]) || (my_face_h[1] != ghost_pinned_buffer_h[1]); // pinned buffers

    if (!initComms || comms_reset) LatticeField::createComms(no_comms_fill);

    if (ghost_field_reset) destroyIPCComms();
    createIPCComms();
  }

  void cudaGaugeField::recvStart(int dim, int dir)
  {
    if (!comm_dim_partitioned(dim)) return;

    if (dir==0) { // sending backwards
      // receive from the processor in the +1 direction
      if (comm_peer2peer_enabled(1,dim)) {
	comm_start(mh_recv_p2p_fwd[bufferIndex][dim]);
      } else if (comm_gdr_enabled()) {
        comm_start(mh_recv_rdma_fwd[bufferIndex][dim]);
      } else {
        comm_start(mh_recv_fwd[bufferIndex][dim]);
      }
    } else { //sending forwards
      // receive from the processor in the -1 direction
      if (comm_peer2peer_enabled(0,dim)) {
	comm_start(mh_recv_p2p_back[bufferIndex][dim]);
      } else if (comm_gdr_enabled()) {
        comm_start(mh_recv_rdma_back[bufferIndex][dim]);
      } else {
        comm_start(mh_recv_back[bufferIndex][dim]);
      }
    }
  }

  void cudaGaugeField::sendStart(int dim, int dir, hipStream_t* stream_p)
  {
    if (!comm_dim_partitioned(dim)) return;

    if (!comm_peer2peer_enabled(dir,dim)) {
      if (dir == 0)
	if (comm_gdr_enabled()) {
	  comm_start(mh_send_rdma_back[bufferIndex][dim]);
	} else {
	  comm_start(mh_send_back[bufferIndex][dim]);
	}
      else
	if (comm_gdr_enabled()) {
	  comm_start(mh_send_rdma_fwd[bufferIndex][dim]);
	} else {
	  comm_start(mh_send_fwd[bufferIndex][dim]);
	}
    } else { // doing peer-to-peer

      void* ghost_dst = static_cast<char*>(ghost_remote_send_buffer_d[bufferIndex][dim][dir])
	+ precision*ghostOffset[dim][(dir+1)%2];

      hipMemcpyAsync(ghost_dst, my_face_dim_dir_d[bufferIndex][dim][dir],
		      ghost_face_bytes[dim], hipMemcpyDeviceToDevice,
		      stream_p ? *stream_p : 0);

      if (dir == 0) {
	// record the event
	hipEventRecord(ipcCopyEvent[bufferIndex][0][dim], stream_p ? *stream_p : 0);
	// send to the processor in the -1 direction
	comm_start(mh_send_p2p_back[bufferIndex][dim]);
      } else {
	hipEventRecord(ipcCopyEvent[bufferIndex][1][dim], stream_p ? *stream_p : 0);
	// send to the processor in the +1 direction
	comm_start(mh_send_p2p_fwd[bufferIndex][dim]);
      }
    }
  }

  void cudaGaugeField::commsComplete(int dim, int dir)
  {
    if (!comm_dim_partitioned(dim)) return;

    if (dir==0) {
      if (comm_peer2peer_enabled(1,dim)) {
	comm_wait(mh_recv_p2p_fwd[bufferIndex][dim]);
	hipEventSynchronize(ipcRemoteCopyEvent[bufferIndex][1][dim]);
      } else if (comm_gdr_enabled()) {
	comm_wait(mh_recv_rdma_fwd[bufferIndex][dim]);
      } else {
	comm_wait(mh_recv_fwd[bufferIndex][dim]);
      }

      if (comm_peer2peer_enabled(0,dim)) {
	comm_wait(mh_send_p2p_back[bufferIndex][dim]);
	hipEventSynchronize(ipcCopyEvent[bufferIndex][0][dim]);
      } else if (comm_gdr_enabled()) {
	comm_wait(mh_send_rdma_back[bufferIndex][dim]);
      } else {
	comm_wait(mh_send_back[bufferIndex][dim]);
      }
    } else {
      if (comm_peer2peer_enabled(0,dim)) {
	comm_wait(mh_recv_p2p_back[bufferIndex][dim]);
	hipEventSynchronize(ipcRemoteCopyEvent[bufferIndex][0][dim]);
      } else if (comm_gdr_enabled()) {
	comm_wait(mh_recv_rdma_back[bufferIndex][dim]);
      } else {
	comm_wait(mh_recv_back[bufferIndex][dim]);
      }

      if (comm_peer2peer_enabled(1,dim)) {
	comm_wait(mh_send_p2p_fwd[bufferIndex][dim]);
	hipEventSynchronize(ipcCopyEvent[bufferIndex][1][dim]);
      } else if (comm_gdr_enabled()) {
	comm_wait(mh_send_rdma_fwd[bufferIndex][dim]);
      } else {
	comm_wait(mh_send_fwd[bufferIndex][dim]);
      }
    }
  }

  void cudaGaugeField::exchangeExtendedGhost(const int *R, bool no_comms_fill)
  {
    const int b = bufferIndex;
    void *send_d[QUDA_MAX_DIM], *recv_d[QUDA_MAX_DIM];

    createComms(R, no_comms_fill);

    size_t offset = 0;
    for (int dim=0; dim<nDim; dim++) {
      if ( !(comm_dim_partitioned(dim) || (no_comms_fill && R[dim])) ) continue;
      send_d[dim] = static_cast<char*>(ghost_send_buffer_d[b]) + offset;
      recv_d[dim] = static_cast<char*>(ghost_recv_buffer_d[b]) + offset;
      offset += 2*ghost_face_bytes[dim]; // factor of two from fwd/back
    }

    for (int dim=0; dim<nDim; dim++) {
      if ( !(comm_dim_partitioned(dim) || (no_comms_fill && R[dim])) ) continue;

      //extract into a contiguous buffer
      extractExtendedGaugeGhost(*this, dim, R, send_d, true);

      if (comm_dim_partitioned(dim)) {
	for (int dir=0; dir<2; dir++) recvStart(dim, dir);

	for (int dir=0; dir<2; dir++) {
	  // issue host-to-device copies if needed
	  if (!comm_peer2peer_enabled(dir,dim) && !comm_gdr_enabled()) {
	    hipMemcpyAsync(my_face_dim_dir_h[bufferIndex][dim][dir], my_face_dim_dir_d[bufferIndex][dim][dir],
			    ghost_face_bytes[dim], hipMemcpyDeviceToHost, streams[dir]);
	  }
	}

	// if neither direction is peer-to-peer then we need to synchronize
	if (!comm_peer2peer_enabled(0,dim) || !comm_peer2peer_enabled(1,dim)) hipDeviceSynchronize();

	// if we pass a stream to sendStart then we must ensure that stream is synchronized
	for (int dir=0; dir<2; dir++) sendStart(dim, dir, &streams[dir]);
	for (int dir=0; dir<2; dir++) commsComplete(dim, dir);

	for (int dir=0; dir<2; dir++) {
	  // issue host-to-device copies if needed
	  if (!comm_peer2peer_enabled(dir,dim) && !comm_gdr_enabled()) {
	    hipMemcpyAsync(from_face_dim_dir_d[bufferIndex][dim][dir], from_face_dim_dir_h[bufferIndex][dim][dir],
			    ghost_face_bytes[dim], hipMemcpyHostToDevice, streams[dir]);
	  }
	}

      } else { // if just doing a local exchange to fill halo then need to swap faces
	qudaMemcpy(from_face_dim_dir_d[b][dim][1], my_face_dim_dir_d[b][dim][0],
		   ghost_face_bytes[dim], hipMemcpyDeviceToDevice);
	qudaMemcpy(from_face_dim_dir_d[b][dim][0], my_face_dim_dir_d[b][dim][1],
		   ghost_face_bytes[dim], hipMemcpyDeviceToDevice);
      }

      // inject back into the gauge field
      extractExtendedGaugeGhost(*this, dim, R, recv_d, false);
    }

    bufferIndex = 1-bufferIndex;
    hipDeviceSynchronize();
  }

  void cudaGaugeField::setGauge(void *gauge_)
  {
    if(create != QUDA_REFERENCE_FIELD_CREATE) {
      errorQuda("Setting gauge pointer is only allowed when create="
          "QUDA_REFERENCE_FIELD_CREATE type\n");
    }
    gauge = gauge_;
  }

  void *create_gauge_buffer(size_t bytes, QudaGaugeFieldOrder order, QudaFieldGeometry geometry) {
    if (order == QUDA_QDP_GAUGE_ORDER) {
      void **buffer = new void*[geometry];
      for (int d=0; d<geometry; d++) buffer[d] = pool_device_malloc(bytes/geometry);
      return ((void*)buffer);
    } else {
      return pool_device_malloc(bytes);
    }

  }

  void **create_ghost_buffer(size_t bytes[], QudaGaugeFieldOrder order, QudaFieldGeometry geometry) {

    if (order > 4) {
      void **buffer = new void*[geometry];
      for (int d=0; d<geometry; d++) buffer[d] = pool_device_malloc(bytes[d]);
      return buffer;
    } else {
      return 0;
    }

  }

  void free_gauge_buffer(void *buffer, QudaGaugeFieldOrder order, QudaFieldGeometry geometry) {
    if (order == QUDA_QDP_GAUGE_ORDER) {
      for (int d=0; d<geometry; d++) pool_device_free(((void**)buffer)[d]);
      delete []((void**)buffer);
    } else {
      pool_device_free(buffer);
    }
  }

  void free_ghost_buffer(void **buffer, QudaGaugeFieldOrder order, QudaFieldGeometry geometry) {
    if (order > 4) {
      for (int d=0; d<geometry; d++) pool_device_free(buffer[d]);
      delete []buffer;
    }
  }

  void cudaGaugeField::copy(const GaugeField &src) {
    if (this == &src) return;

    checkField(src);

    if (link_type == QUDA_ASQTAD_FAT_LINKS) {
      fat_link_max = src.LinkMax();
      if (precision == QUDA_HALF_PRECISION && fat_link_max == 0.0) 
        errorQuda("fat_link_max has not been computed");
    } else {
      fat_link_max = 1.0;
    }

    if (typeid(src) == typeid(cudaGaugeField)) {

      // copy field and ghost zone into this field
      copyGenericGauge(*this, src, QUDA_CUDA_FIELD_LOCATION, gauge, 
          static_cast<const cudaGaugeField&>(src).gauge);

      if (geometry == QUDA_COARSE_GEOMETRY)
	copyGenericGauge(*this, src, QUDA_CUDA_FIELD_LOCATION, gauge, static_cast<const cudaGaugeField&>(src).gauge, 0, 0, 3);

    } else if (typeid(src) == typeid(cpuGaugeField)) {
      if (reorder_location() == QUDA_CPU_FIELD_LOCATION) { // do reorder on the CPU
	void *buffer = pool_pinned_malloc(bytes);

	if (src.GhostExchange() != QUDA_GHOST_EXCHANGE_EXTENDED) {
	  // copy field and ghost zone into buffer
	  copyGenericGauge(*this, src, QUDA_CPU_FIELD_LOCATION, buffer, static_cast<const cpuGaugeField&>(src).gauge);
	} else {
	  copyExtendedGauge(*this, src, QUDA_CPU_FIELD_LOCATION, buffer, static_cast<const cpuGaugeField&>(src).gauge);
	}

	// this copies over both even and odd
	qudaMemcpy(gauge, buffer, bytes, hipMemcpyHostToDevice);
	pool_pinned_free(buffer);
      } else { // else on the GPU

	if (src.Order() == QUDA_MILC_SITE_GAUGE_ORDER) {
	  // special case where we use zero-copy memory to read/write directly from MILC's data
	  void *src_d;
	  hipError_t error = hipHostGetDevicePointer(&src_d, const_cast<void*>(src.Gauge_p()), 0);
	  if (error != hipSuccess) errorQuda("Failed to get device pointer for MILC site array");

	  if (src.GhostExchange() == QUDA_GHOST_EXCHANGE_NO) {
	    copyGenericGauge(*this, src, QUDA_CUDA_FIELD_LOCATION, gauge, src_d);
	  } else {
	    errorQuda("Ghost copy not supported here");
	  }

	} else {
	  void *buffer = create_gauge_buffer(src.Bytes(), src.Order(), src.Geometry());
	  size_t ghost_bytes[8];
	  int srcNinternal = src.Reconstruct() != QUDA_RECONSTRUCT_NO ? src.Reconstruct() : 2*nColor*nColor;
	  for (int d=0; d<geometry; d++) ghost_bytes[d] = nFace * surface[d%4] * srcNinternal * src.Precision();
	  void **ghost_buffer = (nFace > 0) ? create_ghost_buffer(ghost_bytes, src.Order(), geometry) : nullptr;

	  if (src.Order() == QUDA_QDP_GAUGE_ORDER) {
	    for (int d=0; d<geometry; d++) {
	      qudaMemcpy(((void**)buffer)[d], ((void**)src.Gauge_p())[d], src.Bytes()/geometry, hipMemcpyHostToDevice);
	    }
	  } else {
	    qudaMemcpy(buffer, src.Gauge_p(), src.Bytes(), hipMemcpyHostToDevice);
	  }

	  if (src.Order() > 4 && GhostExchange() == QUDA_GHOST_EXCHANGE_PAD &&
	      src.GhostExchange() == QUDA_GHOST_EXCHANGE_PAD && nFace)
	    for (int d=0; d<geometry; d++)
	      qudaMemcpy(ghost_buffer[d], src.Ghost()[d], ghost_bytes[d], hipMemcpyHostToDevice);

	  if (src.GhostExchange() != QUDA_GHOST_EXCHANGE_EXTENDED) {
	    copyGenericGauge(*this, src, QUDA_CUDA_FIELD_LOCATION, gauge, buffer, 0, ghost_buffer);
	    if (geometry == QUDA_COARSE_GEOMETRY) copyGenericGauge(*this, src, QUDA_CUDA_FIELD_LOCATION, gauge, buffer, 0, ghost_buffer, 3);
	  } else {
	    copyExtendedGauge(*this, src, QUDA_CUDA_FIELD_LOCATION, gauge, buffer);
	  }
	  free_gauge_buffer(buffer, src.Order(), src.Geometry());
	  if (nFace > 0) free_ghost_buffer(ghost_buffer, src.Order(), geometry);
	}
      } // reorder_location
    } else {
      errorQuda("Invalid gauge field type");
    }

    // if we have copied from a source without a pad then we need to exchange
    if (ghostExchange == QUDA_GHOST_EXCHANGE_PAD && src.GhostExchange() != QUDA_GHOST_EXCHANGE_PAD)
      exchangeGhost(geometry == QUDA_VECTOR_GEOMETRY ? QUDA_LINK_BACKWARDS : QUDA_LINK_BIDIRECTIONAL);

    staggeredPhaseApplied = src.StaggeredPhaseApplied();
    staggeredPhaseType = src.StaggeredPhase();

    checkCudaError();
  }

  void cudaGaugeField::loadCPUField(const cpuGaugeField &cpu) {
    copy(cpu);
    hipDeviceSynchronize();
    checkCudaError();
  }

  void cudaGaugeField::saveCPUField(cpuGaugeField &cpu) const
  {
    static_cast<LatticeField&>(cpu).checkField(*this);

    if (reorder_location() == QUDA_CUDA_FIELD_LOCATION) {

      if (cpu.Order() == QUDA_MILC_SITE_GAUGE_ORDER) {
	// special case where we use zero-copy memory to read/write directly from MILC's data
	void *cpu_d;
  hipError_t error = hipHostGetDevicePointer(&cpu_d, const_cast<void*>(cpu.Gauge_p()), 0);
  if (error != hipSuccess) errorQuda("Failed to get device pointer for MILC site array");
	if (cpu.GhostExchange() == QUDA_GHOST_EXCHANGE_NO) {
	  copyGenericGauge(cpu, *this, QUDA_CUDA_FIELD_LOCATION, cpu_d, gauge);
	} else {
	  errorQuda("Ghost copy not supported here");
	}
      } else {
	void *buffer = create_gauge_buffer(cpu.Bytes(), cpu.Order(), cpu.Geometry());

	// Allocate space for ghost zone if required
	size_t ghost_bytes[8];
	int cpuNinternal = cpu.Reconstruct() != QUDA_RECONSTRUCT_NO ? cpu.Reconstruct() : 2*nColor*nColor;
	for (int d=0; d<geometry; d++) ghost_bytes[d] = nFace * surface[d%4] * cpuNinternal * cpu.Precision();
	void **ghost_buffer = (nFace > 0) ? create_ghost_buffer(ghost_bytes, cpu.Order(), geometry) : nullptr;

	if (cpu.GhostExchange() != QUDA_GHOST_EXCHANGE_EXTENDED) {
	  copyGenericGauge(cpu, *this, QUDA_CUDA_FIELD_LOCATION, buffer, gauge, ghost_buffer, 0);
	  if (geometry == QUDA_COARSE_GEOMETRY) copyGenericGauge(cpu, *this, QUDA_CUDA_FIELD_LOCATION, buffer, gauge, ghost_buffer, 0, 3);
	} else {
	  copyExtendedGauge(cpu, *this, QUDA_CUDA_FIELD_LOCATION, buffer, gauge);
	}

	if (cpu.Order() == QUDA_QDP_GAUGE_ORDER) {
	  for (int d=0; d<geometry; d++) qudaMemcpy(((void**)cpu.gauge)[d], ((void**)buffer)[d], cpu.Bytes()/geometry, hipMemcpyDeviceToHost);
	} else {
	  qudaMemcpy(cpu.gauge, buffer, cpu.Bytes(), hipMemcpyDeviceToHost);
	}

	if (cpu.Order() > 4 && GhostExchange() == QUDA_GHOST_EXCHANGE_PAD &&
	    cpu.GhostExchange() == QUDA_GHOST_EXCHANGE_PAD && nFace)
	  for (int d=0; d<geometry; d++)
	    qudaMemcpy(cpu.Ghost()[d], ghost_buffer[d], ghost_bytes[d], hipMemcpyDeviceToHost);

	free_gauge_buffer(buffer, cpu.Order(), cpu.Geometry());
	if (nFace > 0) free_ghost_buffer(ghost_buffer, cpu.Order(), geometry);
      }
    } else if (reorder_location() == QUDA_CPU_FIELD_LOCATION) { // do copy then host-side reorder

      void *buffer = pool_pinned_malloc(bytes);
      qudaMemcpy(buffer, gauge, bytes, hipMemcpyDeviceToHost);

      if (cpu.GhostExchange() != QUDA_GHOST_EXCHANGE_EXTENDED) {
	copyGenericGauge(cpu, *this, QUDA_CPU_FIELD_LOCATION, cpu.gauge, buffer);
      } else {
	copyExtendedGauge(cpu, *this, QUDA_CPU_FIELD_LOCATION, cpu.gauge, buffer);
      }
      pool_pinned_free(buffer);

    } else {
      errorQuda("Invalid pack location %d", reorder_location());
    }

    cpu.staggeredPhaseApplied = staggeredPhaseApplied;
    cpu.staggeredPhaseType = staggeredPhaseType;

    hipDeviceSynchronize();
    checkCudaError();
  }

  void cudaGaugeField::backup() const {
    if (backed_up) errorQuda("Gauge field already backed up");
    backup_h = new char[bytes];
    hipMemcpy(backup_h, gauge, bytes, hipMemcpyDeviceToHost);
    checkCudaError();
    backed_up = true;
  }

  void cudaGaugeField::restore() {
    if (!backed_up) errorQuda("Cannot restore since not backed up");
    hipMemcpy(gauge, backup_h, bytes, hipMemcpyHostToDevice);
    delete []backup_h;
    checkCudaError();
    backed_up = false;
  }

  void cudaGaugeField::zero() {
    hipMemset(gauge, 0, bytes);
  }


} // namespace quda
