#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <typeinfo>
#include <string.h>
#include <iostream>

#include <color_spinor_field.h>
#include <color_spinor_field_order.h> // Evan hack
#include <blas_quda.h>
#include <dslash_quda.h>

int zeroCopy = 0;

namespace quda {

  static bool ghost_precision_reset = false;
  static QudaPrecision ghost_precision_old = QUDA_INVALID_PRECISION;

  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorParam &param) : 
    ColorSpinorField(param), alloc(false), init(true), texInit(false),
    ghostTexInit(false), ghost_field_tex{nullptr,nullptr,nullptr,nullptr}, bufferMessageHandler(0)
  {
    // this must come before create
    if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      v = param.v;
      norm = param.norm;
    }

    create(param.create);

    if  (param.create == QUDA_NULL_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_ZERO_FIELD_CREATE) {
      zero();
    } else if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_COPY_FIELD_CREATE) {
      errorQuda("not implemented");
    }
  }

  cudaColorSpinorField::cudaColorSpinorField(const cudaColorSpinorField &src) : 
    ColorSpinorField(src), alloc(false), init(true), texInit(false),
    ghostTexInit(false), ghost_field_tex{nullptr,nullptr,nullptr,nullptr}, bufferMessageHandler(0)
  {
    create(QUDA_COPY_FIELD_CREATE);
    copySpinorField(src);
  }

  // creates a copy of src, any differences defined in param
  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorField &src, 
					     const ColorSpinorParam &param) :
    ColorSpinorField(src), alloc(false), init(true), texInit(false),
    ghostTexInit(false), ghost_field_tex{nullptr,nullptr,nullptr,nullptr}, bufferMessageHandler(0)
  {
    // can only overide if we are not using a reference or parity special case
    if (param.create != QUDA_REFERENCE_FIELD_CREATE || 
	(param.create == QUDA_REFERENCE_FIELD_CREATE && 
	 src.SiteSubset() == QUDA_FULL_SITE_SUBSET && 
	 param.siteSubset == QUDA_PARITY_SITE_SUBSET && 
	 typeid(src) == typeid(cudaColorSpinorField) ) || 
         (param.create == QUDA_REFERENCE_FIELD_CREATE && (param.is_composite || param.is_component))) {
      reset(param);
    } else {
      errorQuda("Undefined behaviour"); // else silent bug possible?
    }

    // This must be set before create is called
    if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      if (typeid(src) == typeid(cudaColorSpinorField)) {
	v = (void*)src.V();
	norm = (void*)src.Norm();
      } else {
	errorQuda("Cannot reference a non-cuda field");
      }

      if (composite_descr.is_component && !(src.SiteSubset() == QUDA_FULL_SITE_SUBSET && this->SiteSubset() == QUDA_PARITY_SITE_SUBSET)) 
      {//setup eigenvector form the set
        v    = (void*)((char*)v    + composite_descr.id*bytes);         
        norm = (void*)((char*)norm + composite_descr.id*norm_bytes);         
      }
    }

    create(param.create);

    if (param.create == QUDA_NULL_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_ZERO_FIELD_CREATE) {
      zero();
    } else if (param.create == QUDA_COPY_FIELD_CREATE) {
      copySpinorField(src);
    } else if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      // do nothing
    } else {
      errorQuda("CreateType %d not implemented", param.create);
    }

  }

  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorField &src) 
    : ColorSpinorField(src), alloc(false), init(true), texInit(false),
      ghostTexInit(false), ghost_field_tex{nullptr,nullptr,nullptr,nullptr}, bufferMessageHandler(0)
  {
    create(QUDA_COPY_FIELD_CREATE);
    copySpinorField(src);
  }

  ColorSpinorField& cudaColorSpinorField::operator=(const ColorSpinorField &src) {
    if (typeid(src) == typeid(cudaColorSpinorField)) {
      *this = (dynamic_cast<const cudaColorSpinorField&>(src));
    } else if (typeid(src) == typeid(cpuColorSpinorField)) {
      *this = (dynamic_cast<const cpuColorSpinorField&>(src));
    } else {
      errorQuda("Unknown input ColorSpinorField %s", typeid(src).name());
    }
    return *this;
  }

  cudaColorSpinorField& cudaColorSpinorField::operator=(const cudaColorSpinorField &src) {
    if (&src != this) {
      // keep current attributes unless unset
      if (!ColorSpinorField::init) { // note this will turn a reference field into a regular field
	destroy();
	destroyComms(); // not sure if this necessary
	ColorSpinorField::operator=(src);
	create(QUDA_COPY_FIELD_CREATE);
      }
      copySpinorField(src);
    }
    return *this;
  }

  cudaColorSpinorField& cudaColorSpinorField::operator=(const cpuColorSpinorField &src) {
    // keep current attributes unless unset
    if (!ColorSpinorField::init) { // note this will turn a reference field into a regular field
      destroy();
      ColorSpinorField::operator=(src);
      create(QUDA_COPY_FIELD_CREATE);
    }
    loadSpinorField(src);
    return *this;
  }

  cudaColorSpinorField::~cudaColorSpinorField() {
    destroyComms();
    destroy();
  }

  void cudaColorSpinorField::create(const QudaFieldCreate create) {

    if (siteSubset == QUDA_FULL_SITE_SUBSET && siteOrder != QUDA_EVEN_ODD_SITE_ORDER) {
      errorQuda("Subset not implemented");
    }

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      switch(mem_type) {
      case QUDA_MEMORY_DEVICE:
	v = pool_device_malloc(bytes);
	if (precision == QUDA_HALF_PRECISION || precision == QUDA_QUARTER_PRECISION) norm = pool_device_malloc(norm_bytes);
	break;
      case QUDA_MEMORY_MAPPED:
	v_h = mapped_malloc(bytes);
	hipHostGetDevicePointer(&v, v_h, 0); // set the matching device pointer
	if (precision == QUDA_HALF_PRECISION || precision == QUDA_QUARTER_PRECISION) {
	  norm_h = mapped_malloc(norm_bytes);
	  hipHostGetDevicePointer(&norm, norm_h, 0); // set the matching device pointer
	}
	break;
      default:
	errorQuda("Unsupported memory type %d", mem_type);
      }
      alloc = true;
    }

    if (siteSubset == QUDA_FULL_SITE_SUBSET) {
      if(composite_descr.is_composite && (create != QUDA_REFERENCE_FIELD_CREATE)) {
	if(composite_descr.dim <= 0) errorQuda("\nComposite size is not defined\n");
	  
        ColorSpinorParam param;
        param.siteSubset = QUDA_FULL_SITE_SUBSET;
        param.nDim = nDim;
        memcpy(param.x, x, nDim*sizeof(int));
        param.create = QUDA_REFERENCE_FIELD_CREATE;
        param.v = v;
        param.norm = norm;
        param.is_composite   = false;
        param.composite_dim  = 0;
        param.is_component = true;
	param.mem_type = mem_type;

        components.reserve(composite_descr.dim);
        for(int cid = 0; cid < composite_descr.dim; cid++) {
	  param.component_id = cid;
	  components.push_back(new cudaColorSpinorField(*this, param));
        }
      } else {
        // create the associated even and odd subsets
        ColorSpinorParam param;
        param.siteSubset = QUDA_PARITY_SITE_SUBSET;
        param.nDim = nDim;
        memcpy(param.x, x, nDim*sizeof(int));
        param.x[0] /= 2; // set single parity dimensions
        param.create = QUDA_REFERENCE_FIELD_CREATE;
        param.v = v;
        param.norm = norm;
        param.is_composite  = false;
        param.composite_dim = 0;
        param.is_component  = composite_descr.is_component;
        param.component_id  = composite_descr.id;
	param.mem_type = mem_type;

        even = new cudaColorSpinorField(*this, param);
        odd = new cudaColorSpinorField(*this, param);

        // need this hackery for the moment (need to locate the odd pointers half way into the full field)
        (dynamic_cast<cudaColorSpinorField*>(odd))->v = (void*)((char*)v + bytes/2);
        if (precision == QUDA_HALF_PRECISION || precision == QUDA_QUARTER_PRECISION) 
	  (dynamic_cast<cudaColorSpinorField*>(odd))->norm = (void*)((char*)norm + norm_bytes/2);

#ifdef USE_TEXTURE_OBJECTS
        dynamic_cast<cudaColorSpinorField*>(even)->destroyTexObject();
        dynamic_cast<cudaColorSpinorField*>(even)->createTexObject();
        dynamic_cast<cudaColorSpinorField*>(odd)->destroyTexObject();
        dynamic_cast<cudaColorSpinorField*>(odd)->createTexObject();
#endif
      }
    } else { //siteSubset == QUDA_PARITY_SITE_SUBSET

      //! setup an object for selected eigenvector (the 1st one as a default):
      if (composite_descr.is_composite && (create != QUDA_REFERENCE_FIELD_CREATE)) 
      {
         if(composite_descr.dim <= 0) errorQuda("\nComposite size is not defined\n");
         //if(bytes > 1811939328) warningQuda("\nCUDA API probably won't be able to create texture object for the eigenvector set... Object size is : %u bytes\n", bytes);
         if (getVerbosity() == QUDA_DEBUG_VERBOSE) printfQuda("\nEigenvector set constructor...\n");
         // create the associated even and odd subsets
         ColorSpinorParam param;
         param.siteSubset = QUDA_PARITY_SITE_SUBSET;
         param.nDim = nDim;
         memcpy(param.x, x, nDim*sizeof(int));
         param.create = QUDA_REFERENCE_FIELD_CREATE;
         param.v = v;
         param.norm = norm;
         param.is_composite   = false;
         param.composite_dim  = 0;
         param.is_component = true;
	 param.mem_type = mem_type;

         //reserve eigvector set
         components.reserve(composite_descr.dim);
         //setup volume, [real_]length and stride for a single eigenvector
         for(int cid = 0; cid < composite_descr.dim; cid++)
         {
            param.component_id = cid;
            components.push_back(new cudaColorSpinorField(*this, param));

#ifdef USE_TEXTURE_OBJECTS //(a lot of texture objects...)
            dynamic_cast<cudaColorSpinorField*>(components[cid])->destroyTexObject();
            dynamic_cast<cudaColorSpinorField*>(components[cid])->createTexObject();
#endif
         }
      }
    }

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      if (siteSubset != QUDA_FULL_SITE_SUBSET) {
	zeroPad();
      } else if(!composite_descr.is_composite) {
	(dynamic_cast<cudaColorSpinorField*>(even))->zeroPad();
	(dynamic_cast<cudaColorSpinorField*>(odd))->zeroPad();
      } else { //temporary hack for the full spinor field sets, manual zeroPad for each component:
	for(int cid = 0; cid < composite_descr.dim; cid++) {
	  (dynamic_cast<cudaColorSpinorField&>(components[cid]->Even())).zeroPad();
	  (dynamic_cast<cudaColorSpinorField&>(components[cid]->Odd())).zeroPad();
	}
      }
    }

#ifdef USE_TEXTURE_OBJECTS
    if (!composite_descr.is_composite || composite_descr.is_component)
      createTexObject();
#endif
  }

#ifdef USE_TEXTURE_OBJECTS
  void cudaColorSpinorField::createTexObject() {

    if ( (isNative() || fieldOrder == QUDA_FLOAT2_FIELD_ORDER) && nVec == 1 ) {
      if (texInit) errorQuda("Already bound textures");
      
      // create the texture for the field components
      
      hipChannelFormatDesc desc;
      memset(&desc, 0, sizeof(hipChannelFormatDesc));
      if (precision == QUDA_SINGLE_PRECISION) desc.f = hipChannelFormatKindFloat;
      else desc.f = hipChannelFormatKindSigned; // quarter is char, half is short, double is int2
      
      // staggered and coarse fields in half and single are always two component
      int texel_size = 1;
      // all FLOAT2-ordred fields that are not double precision
      if (precision != QUDA_DOUBLE_PRECISION && fieldOrder == QUDA_FLOAT2_FIELD_ORDER) {
        desc.x = 8*precision;
        desc.y = 8*precision;
        desc.z = 0;
        desc.w = 0;
        texel_size = 2*precision;
      } else { // all others are four component (double2 is spread across int4)
        desc.x = (precision == QUDA_DOUBLE_PRECISION) ? 8*sizeof(int) : 8*precision;
        desc.y = (precision == QUDA_DOUBLE_PRECISION) ? 8*sizeof(int) : 8*precision;
        desc.z = (precision == QUDA_DOUBLE_PRECISION) ? 8*sizeof(int) : 8*precision;
        desc.w = (precision == QUDA_DOUBLE_PRECISION) ? 8*sizeof(int) : 8*precision;
        texel_size = 4 * (precision == QUDA_DOUBLE_PRECISION ? sizeof(int) : precision);
      }
      
      hipResourceDesc resDesc;
      memset(&resDesc, 0, sizeof(resDesc));
      resDesc.resType = hipResourceTypeLinear;
      resDesc.res.linear.devPtr = v;
      resDesc.res.linear.desc = desc;
      resDesc.res.linear.sizeInBytes = bytes;
      
      hipTextureDesc texDesc;
      memset(&texDesc, 0, sizeof(texDesc));
      if (precision == QUDA_HALF_PRECISION || precision == QUDA_QUARTER_PRECISION) texDesc.readMode = hipReadModeNormalizedFloat;
      else texDesc.readMode = hipReadModeElementType;

      if (resDesc.res.linear.sizeInBytes % deviceProp.textureAlignment != 0) {
        errorQuda("Allocation size %lu does not have correct alignment for textures (%lu)",
          resDesc.res.linear.sizeInBytes, deviceProp.textureAlignment);
      }

      unsigned long texels = resDesc.res.linear.sizeInBytes / texel_size;
      if (texels > (unsigned)deviceProp.maxTexture1DLinear) {
        errorQuda("Attempting to bind too large a texture %lu > %d", texels, deviceProp.maxTexture1DLinear);
      }

      hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

      checkCudaError();

      // create the texture for the norm components
      if (precision == QUDA_HALF_PRECISION || precision == QUDA_QUARTER_PRECISION) {
        hipChannelFormatDesc desc;
        memset(&desc, 0, sizeof(hipChannelFormatDesc));
        desc.f = hipChannelFormatKindFloat;
        desc.x = 8*QUDA_SINGLE_PRECISION; desc.y = 0; desc.z = 0; desc.w = 0;

        hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypeLinear;
        resDesc.res.linear.devPtr = norm;
        resDesc.res.linear.desc = desc;
        resDesc.res.linear.sizeInBytes = norm_bytes;

        if (resDesc.res.linear.sizeInBytes % deviceProp.textureAlignment != 0) {
          errorQuda("Allocation size %lu does not have correct alignment for textures (%lu)",
        	    resDesc.res.linear.sizeInBytes, deviceProp.textureAlignment);
        }

        hipTextureDesc texDesc;
        memset(&texDesc, 0, sizeof(texDesc));
        texDesc.readMode = hipReadModeElementType;

        hipCreateTextureObject(&texNorm, &resDesc, &texDesc, NULL);

        checkCudaError();
      }
      
      texInit = true;

      checkCudaError();
    }
  }

  void cudaColorSpinorField::createGhostTexObject() const {
    // create the ghost texture object
    if ( (isNative() || fieldOrder == QUDA_FLOAT2_FIELD_ORDER) && nVec == 1 && ghost_bytes) {
      if (ghostTexInit) errorQuda("Already bound ghost texture");

      for (int b=0; b<2; b++) {
	hipChannelFormatDesc desc;
	memset(&desc, 0, sizeof(hipChannelFormatDesc));
	if (ghost_precision == QUDA_SINGLE_PRECISION) desc.f = hipChannelFormatKindFloat;
	else desc.f = hipChannelFormatKindSigned; // half is short, double is int2

	// all FLOAT2-ordred fields that are not double precision
	if (ghost_precision != QUDA_DOUBLE_PRECISION && fieldOrder == QUDA_FLOAT2_FIELD_ORDER) {
	  desc.x = 8*ghost_precision;
	  desc.y = 8*ghost_precision;
	  desc.z = 0;
	  desc.w = 0;
	} else { // all others are four component (double2 is spread across int4)
	  desc.x = (ghost_precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*ghost_precision;
	  desc.y = (ghost_precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*ghost_precision;
	  desc.z = (ghost_precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*ghost_precision;
	  desc.w = (ghost_precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*ghost_precision;
	}

	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = ghost_recv_buffer_d[b];
	resDesc.res.linear.desc = desc;
	resDesc.res.linear.sizeInBytes = ghost_bytes;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	if (ghost_precision == QUDA_HALF_PRECISION || ghost_precision == QUDA_QUARTER_PRECISION) texDesc.readMode = hipReadModeNormalizedFloat;
	else texDesc.readMode = hipReadModeElementType;

	hipCreateTextureObject(&ghostTex[b], &resDesc, &texDesc, NULL);

	// second set of ghost texture map to the host-mapped pinned receive buffers
	resDesc.res.linear.devPtr = static_cast<char*>(ghost_pinned_buffer_hd[b])+ghost_bytes;
	hipCreateTextureObject(&ghostTex[2+b], &resDesc, &texDesc, NULL);

	if (ghost_precision == QUDA_HALF_PRECISION || ghost_precision == QUDA_QUARTER_PRECISION) {
	  hipChannelFormatDesc desc;
	  memset(&desc, 0, sizeof(hipChannelFormatDesc));
	  desc.f = hipChannelFormatKindFloat;
	  desc.x = 8*QUDA_SINGLE_PRECISION; desc.y = 0; desc.z = 0; desc.w = 0;

	  hipResourceDesc resDesc;
	  memset(&resDesc, 0, sizeof(resDesc));
	  resDesc.resType = hipResourceTypeLinear;
	  resDesc.res.linear.devPtr = ghost_recv_buffer_d[b];
	  resDesc.res.linear.desc = desc;
	  resDesc.res.linear.sizeInBytes = ghost_bytes;

	  hipTextureDesc texDesc;
	  memset(&texDesc, 0, sizeof(texDesc));
	  texDesc.readMode = hipReadModeElementType;

	  hipCreateTextureObject(&ghostTexNorm[b], &resDesc, &texDesc, NULL);

	  resDesc.res.linear.devPtr = static_cast<char*>(ghost_pinned_buffer_hd[b])+ghost_bytes;
	  hipCreateTextureObject(&ghostTexNorm[2+b], &resDesc, &texDesc, NULL);
	}

	ghost_field_tex[b] = ghost_recv_buffer_d[b];
	ghost_field_tex[2+b] = static_cast<char*>(ghost_pinned_buffer_hd[b])+ghost_bytes;
      } // buffer index

      ghostTexInit = true;

      checkCudaError();
    }

  }

  void cudaColorSpinorField::destroyTexObject() {
    if ( (isNative() || fieldOrder == QUDA_FLOAT2_FIELD_ORDER) && nVec == 1 && texInit) {
      hipDestroyTextureObject(tex);
      if (precision == QUDA_HALF_PRECISION || precision == QUDA_QUARTER_PRECISION) hipDestroyTextureObject(texNorm);
      texInit = false;
    }
  }

  void cudaColorSpinorField::destroyGhostTexObject() const {
    if ( (isNative() || fieldOrder == QUDA_FLOAT2_FIELD_ORDER) && nVec == 1 && ghostTexInit) {
      for (int i=0; i<4; i++) hipDestroyTextureObject(ghostTex[i]);
      if ( (ghost_precision_reset && (ghost_precision_old == QUDA_HALF_PRECISION || ghost_precision_old == QUDA_QUARTER_PRECISION) ||
        (!ghost_precision_reset && (ghost_precision == QUDA_HALF_PRECISION || ghost_precision == QUDA_QUARTER_PRECISION) ) ) ) {
        for (int i=0; i<4; i++) hipDestroyTextureObject(ghostTexNorm[i]);
      }
      ghostTexInit = false;
    }
  }
#endif

  void cudaColorSpinorField::destroy() {

    if (alloc) {
      switch(mem_type) {
      case QUDA_MEMORY_DEVICE:
        pool_device_free(v);
        if (precision == QUDA_HALF_PRECISION || precision == QUDA_QUARTER_PRECISION) pool_device_free(norm);
        break;
      case QUDA_MEMORY_MAPPED:
        host_free(v_h);
        if (precision == QUDA_HALF_PRECISION || precision == QUDA_QUARTER_PRECISION) host_free(norm_h);
        break;
      default:
        errorQuda("Unsupported memory type %d", mem_type);
      }
    }


    if (composite_descr.is_composite) 
    {
       CompositeColorSpinorField::iterator vec;
       for (vec = components.begin(); vec != components.end(); vec++) delete *vec;
    } 

    if ( siteSubset == QUDA_FULL_SITE_SUBSET && (!composite_descr.is_composite || composite_descr.is_component) ) {
      delete even;
      delete odd;
    }

#ifdef USE_TEXTURE_OBJECTS
    if (!composite_descr.is_composite || composite_descr.is_component) {
      destroyTexObject();
      destroyGhostTexObject();
    }
#endif

  }

  void cudaColorSpinorField::backup() const {
    if (backed_up) errorQuda("ColorSpinorField already backed up");
    backup_h = new char[bytes];
    hipMemcpy(backup_h, v, bytes, hipMemcpyDeviceToHost);
    if (norm_bytes) {
      backup_norm_h = new char[norm_bytes];
      hipMemcpy(backup_norm_h, norm, norm_bytes, hipMemcpyDeviceToHost);
    }
    checkCudaError();
    backed_up = true;
  }

  void cudaColorSpinorField::restore() {
    if (!backed_up) errorQuda("Cannot restore since not backed up");
    hipMemcpy(v, backup_h, bytes, hipMemcpyHostToDevice);
    delete []backup_h;
    if (norm_bytes) {
      hipMemcpy(v, backup_norm_h, norm_bytes, hipMemcpyHostToDevice);
      delete []backup_norm_h;
    }
    checkCudaError();
    backed_up = false;
  }

  // cuda's floating point format, IEEE-754, represents the floating point
  // zero as 4 zero bytes
  void cudaColorSpinorField::zero() {
    hipMemsetAsync(v, 0, bytes, streams[Nstream-1]);
    if (precision == QUDA_HALF_PRECISION || precision == QUDA_QUARTER_PRECISION) hipMemsetAsync(norm, 0, norm_bytes, streams[Nstream-1]);
  }

  void cudaColorSpinorField::zeroPad() {
    size_t pad_bytes = (stride - volume) * precision * fieldOrder;
    int Npad = nColor * nSpin * 2 / fieldOrder;

    if (composite_descr.is_composite && !composite_descr.is_component){//we consider the whole eigenvector set:
      Npad      *= composite_descr.dim;
      pad_bytes /= composite_descr.dim;
    }

    size_t pitch = ((!composite_descr.is_composite || composite_descr.is_component) ? stride : composite_descr.stride)*fieldOrder*precision;
    char   *dst  = (char*)v + ((!composite_descr.is_composite || composite_descr.is_component) ? volume : composite_descr.volume)*fieldOrder*precision;
    if (pad_bytes) hipMemset2D(dst, pitch, 0, pad_bytes, Npad);

    //for (int i=0; i<Npad; i++) {
    //  if (pad_bytes) hipMemset((char*)v + (volume + i*stride)*fieldOrder*precision, 0, pad_bytes);
    //}
  }

  void cudaColorSpinorField::copy(const cudaColorSpinorField &src) {
    checkField(*this, src);
    if (this->GammaBasis() != src.GammaBasis()) errorQuda("cannot call this copy with different basis");
    blas::copy(*this, src);
  }

  void cudaColorSpinorField::copySpinorField(const ColorSpinorField &src) {
    
    // src is on the device and is native
    if (typeid(src) == typeid(cudaColorSpinorField) && 
	isNative() && dynamic_cast<const cudaColorSpinorField &>(src).isNative() &&
	this->GammaBasis() == src.GammaBasis()) {
      copy(dynamic_cast<const cudaColorSpinorField&>(src));
    } else if (typeid(src) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION);
    } else if (typeid(src) == typeid(cpuColorSpinorField)) { // src is on the host
      loadSpinorField(src);
    } else {
      errorQuda("Unknown input ColorSpinorField %s", typeid(src).name());
    }
  } 

  void cudaColorSpinorField::loadSpinorField(const ColorSpinorField &src) {

    if (reorder_location() == QUDA_CPU_FIELD_LOCATION &&typeid(src) == typeid(cpuColorSpinorField)) {
      void *buffer = pool_pinned_malloc(bytes + norm_bytes);
      memset(buffer, 0, bytes+norm_bytes); // FIXME (temporary?) bug fix for padding

      copyGenericColorSpinor(*this, src, QUDA_CPU_FIELD_LOCATION, buffer, 0, static_cast<char*>(buffer)+bytes, 0);

      qudaMemcpy(v, buffer, bytes, hipMemcpyHostToDevice);
      qudaMemcpy(norm, static_cast<char*>(buffer)+bytes, norm_bytes, hipMemcpyHostToDevice);

      pool_pinned_free(buffer);
    } else if (typeid(src) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION);
    } else {
      void *Src=nullptr, *srcNorm=nullptr, *buffer=nullptr;
      if (!zeroCopy) {
	buffer = pool_device_malloc(src.Bytes()+src.NormBytes());
	Src = buffer;
	srcNorm = static_cast<char*>(Src) + src.Bytes();
	qudaMemcpy(Src, src.V(), src.Bytes(), hipMemcpyHostToDevice);
	qudaMemcpy(srcNorm, src.Norm(), src.NormBytes(), hipMemcpyHostToDevice);
      } else {
	buffer = pool_pinned_malloc(src.Bytes()+src.NormBytes());
	memcpy(buffer, src.V(), src.Bytes());
	memcpy(static_cast<char*>(buffer)+src.Bytes(), src.Norm(), src.NormBytes());
	hipHostGetDevicePointer(&Src, buffer, 0);
	srcNorm = static_cast<char*>(Src) + src.Bytes();
      }

      hipMemset(v, 0, bytes); // FIXME (temporary?) bug fix for padding
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION, 0, Src, 0, srcNorm);

      if (zeroCopy) pool_pinned_free(buffer);
      else pool_device_free(buffer);
    }

    return;
  }


  void cudaColorSpinorField::saveSpinorField(ColorSpinorField &dest) const {
    if (reorder_location() == QUDA_CPU_FIELD_LOCATION && typeid(dest) == typeid(cpuColorSpinorField)) {
      void *buffer = pool_pinned_malloc(bytes+norm_bytes);
      qudaMemcpy(buffer, v, bytes, hipMemcpyDeviceToHost);
      qudaMemcpy(static_cast<char*>(buffer)+bytes, norm, norm_bytes, hipMemcpyDeviceToHost);

      copyGenericColorSpinor(dest, *this, QUDA_CPU_FIELD_LOCATION, 0, buffer, 0, static_cast<char*>(buffer)+bytes);
      pool_pinned_free(buffer);
    } else if (typeid(dest) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(dest, *this, QUDA_CUDA_FIELD_LOCATION);
    } else {
      void *dst=nullptr, *dstNorm=nullptr, *buffer=nullptr, *vNorm=nullptr;
      if (!zeroCopy) {
	buffer = pool_device_malloc(dest.Bytes()+dest.NormBytes());
	dst = buffer;
	dstNorm = static_cast<char*>(dst) + dest.Bytes();
      } else {
	buffer = pool_pinned_malloc(dest.Bytes()+dest.NormBytes());
	hipHostGetDevicePointer(&dst, buffer, 0);
	dstNorm = static_cast<char*>(dst)+dest.Bytes();
      }
      vNorm = static_cast<char*>(v)+this->Bytes();
      copyGenericColorSpinor(dest, *this, QUDA_CUDA_FIELD_LOCATION, dst, v, dstNorm, vNorm);

      if (!zeroCopy) {
	qudaMemcpy(dest.V(), dst, dest.Bytes(), hipMemcpyDeviceToHost);
	qudaMemcpy(dest.Norm(), dstNorm, dest.NormBytes(), hipMemcpyDeviceToHost);
      } else {
	memcpy(dest.V(), buffer, dest.Bytes());
	memcpy(dest.Norm(), static_cast<char*>(buffer) + dest.Bytes(), dest.NormBytes());
      }

      if (zeroCopy) pool_pinned_free(buffer);
      else pool_device_free(buffer);
    }

    return;
  }

  void cudaColorSpinorField::allocateGhostBuffer(int nFace, bool spin_project) const {

    createGhostZone(nFace, spin_project);
    LatticeField::allocateGhostBuffer(ghost_bytes);

#ifdef USE_TEXTURE_OBJECTS
    // ghost texture is per object
    if (ghost_field_tex[0] != ghost_recv_buffer_d[0] || ghost_field_tex[1] != ghost_recv_buffer_d[1] || ghost_precision_reset)
      destroyGhostTexObject();
    if (!ghostTexInit) createGhostTexObject();
#endif
  }

  // pack the ghost zone into a contiguous buffer for communications
  void cudaColorSpinorField::packGhost(const int nFace, const QudaParity parity, 
                                       const int dim, const QudaDirection dir,
				       const int dagger, hipStream_t *stream, 
				       MemoryLocation location [2*QUDA_MAX_DIM],
                                       MemoryLocation location_label, double a, double b)
  {
#ifdef MULTI_GPU
    int face_num = (dir == QUDA_BACKWARDS) ? 0 : (dir == QUDA_FORWARDS) ? 1 : 2;
    void *packBuffer[2*QUDA_MAX_DIM];

    for (int dim=0; dim<4; dim++) {
      for (int dir=0; dir<2; dir++) {
	switch(location[2*dim+dir]) {
	case Device: // pack to local device buffer
	  packBuffer[2*dim+dir] = my_face_dim_dir_d[bufferIndex][dim][dir];
          break;
	case Host:   // pack to zero-copy memory
	  packBuffer[2*dim+dir] = my_face_dim_dir_hd[bufferIndex][dim][dir];
          break;
	case Remote:   // pack to remote peer memory
	  packBuffer[2*dim+dir] = static_cast<char*>(ghost_remote_send_buffer_d[bufferIndex][dim][dir]) + precision*ghostOffset[dim][1-dir];
          break;
	default: errorQuda("Undefined location %d", location[2*dim+dir]);
	}
      }
    }

    packFace(packBuffer, *this, location_label, nFace, dagger, parity, dim, face_num, *stream, a, b);
#else
    errorQuda("packGhost not built on single-GPU build");
#endif
  }
 
  // send the ghost zone to the host
  void cudaColorSpinorField::sendGhost(void *ghost_spinor, const int nFace, const int dim, 
				       const QudaDirection dir, const int dagger, 
				       hipStream_t *stream) {

#ifdef MULTI_GPU
    int Nvec = (nSpin == 1 || ghost_precision == QUDA_DOUBLE_PRECISION) ? 2 : 4;
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1);  // (spin proj.) degrees of freedom
    int Npad = Nint / Nvec; // number Nvec buffers we have

    if (precision != ghost_precision) { pushKernelPackT(true); }
    
    if (dim !=3 || getKernelPackT()) { // use kernels to pack into contiguous buffers then a single hipMemcpy

      size_t bytes = nFace*Nint*ghostFace[dim]*ghost_precision;

      if (ghost_precision == QUDA_HALF_PRECISION || ghost_precision == QUDA_QUARTER_PRECISION) bytes += nFace*ghostFace[dim]*sizeof(float);

      void* gpu_buf = (dir == QUDA_BACKWARDS) ? my_face_dim_dir_d[bufferIndex][dim][0] : my_face_dim_dir_d[bufferIndex][dim][1];

      qudaMemcpyAsync(ghost_spinor, gpu_buf, bytes, hipMemcpyDeviceToHost, *stream);

    } else if (this->TwistFlavor() != QUDA_TWIST_NONDEG_DOUBLET) { // do multiple cudaMemcpys

      const int x4 = nDim==5 ? x[4] : 1;
      const int Nt_minus1_offset = (volumeCB - nFace*ghostFace[3])/x4; // N_t -1 = Vh-Vsh

      int offset = 0;
      if (nSpin == 1) {
	offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset;
      } else if (nSpin == 4) {
	// !dagger: send lower components backwards, send upper components forwards
	// dagger: send upper components backwards, send lower components forwards
	bool upper = dagger ? true : false; // Fwd is !Back  
	if (dir == QUDA_FORWARDS) upper = !upper;
	int lower_spin_offset = Npad*stride;
	if (upper) offset = (dir == QUDA_BACKWARDS ? 0 : Nt_minus1_offset);
	else offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? 0 : Nt_minus1_offset);
      }
    
      size_t len = nFace*(ghostFace[3]/x4)*Nvec*ghost_precision;
      size_t dpitch = x4*len;
      size_t spitch = stride*Nvec*ghost_precision;

      // QUDA Memcpy NPad's worth. 
      //  -- Dest will point to the right beginning PAD. 
      //  -- Each Pad has size Nvec*Vsh Floats. 
      //  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next
      for (int s=0; s<x4; s++) { // loop over multiple 4-d volumes (if they exist)
	void *dst = (char*)ghost_spinor + s*len;
	void *src = (char*)v + (offset + s*(volumeCB/x4))*Nvec*ghost_precision;
	qudaMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);

	if (ghost_precision == QUDA_HALF_PRECISION || ghost_precision == QUDA_QUARTER_PRECISION) {
	  size_t len = nFace*(ghostFace[3]/x4)*sizeof(float);
	  int norm_offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset*sizeof(float);
	  void *dst = (char*)ghost_spinor + nFace*Nint*ghostFace[3]*ghost_precision + s*len;
	  void *src = (char*)norm + norm_offset + s*(volumeCB/x4)*sizeof(float);
          qudaMemcpyAsync(dst, src, len, hipMemcpyDeviceToHost, *stream);
	}
      }
    }else{
      int flavorVolume = volume / 2;
      int flavorTFace  = ghostFace[3] / 2;
      int flavor1_Nt_minus1_offset = (flavorVolume - flavorTFace);
      int flavor2_Nt_minus1_offset = (volume - flavorTFace);
      int flavor1_offset = 0;
      int flavor2_offset = 0;
      // !dagger: send lower components backwards, send upper components forwards
      // dagger: send upper components backwards, send lower components forwards
      bool upper = dagger ? true : false; // Fwd is !Back
      if (dir == QUDA_FORWARDS) upper = !upper;
      int lower_spin_offset = Npad*stride;//ndeg tm: stride=2*flavor_volume+pad
      if (upper) {
        flavor1_offset = (dir == QUDA_BACKWARDS ? 0 : flavor1_Nt_minus1_offset);
        flavor2_offset = (dir == QUDA_BACKWARDS ? flavorVolume : flavor2_Nt_minus1_offset);
      }else{
        flavor1_offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? 0 : flavor1_Nt_minus1_offset);
        flavor2_offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? flavorVolume : flavor2_Nt_minus1_offset);
      }

      // QUDA Memcpy NPad's worth.
      //  -- Dest will point to the right beginning PAD.
      //  -- Each Pad has size Nvec*Vsh Floats.
      //  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next

      void *dst = (char*)ghost_spinor;
      void *src = (char*)v + flavor1_offset*Nvec*ghost_precision;
      size_t len = flavorTFace*Nvec*ghost_precision;
      size_t spitch = stride*Nvec*ghost_precision;//ndeg tm: stride=2*flavor_volume+pad
      size_t dpitch = 2*len;
      qudaMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);
      dst = (char*)ghost_spinor+len;
      src = (char*)v + flavor2_offset*Nvec*ghost_precision;
      qudaMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);

      if (ghost_precision == QUDA_HALF_PRECISION || ghost_precision == QUDA_QUARTER_PRECISION) {
        int Nt_minus1_offset = (flavorVolume - flavorTFace);
        int norm_offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset*sizeof(float);
	void *dst = (char*)ghost_spinor + Nint*ghostFace[3]*ghost_precision;
	void *src = (char*)norm + norm_offset;
        size_t dpitch = flavorTFace*sizeof(float);
        size_t spitch = flavorVolume*sizeof(float);
	qudaMemcpy2DAsync(dst, dpitch, src, spitch, flavorTFace*sizeof(float), 2, hipMemcpyDeviceToHost, *stream);
      }
    }

    if (precision != ghost_precision) { popKernelPackT(); }

#else
    errorQuda("sendGhost not built on single-GPU build");
#endif

  }


  void cudaColorSpinorField::unpackGhost(const void* ghost_spinor, const int nFace, 
					 const int dim, const QudaDirection dir, 
					 const int dagger, hipStream_t* stream) 
  {
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1);  // (spin proj.) degrees of freedom

    int len = nFace*ghostFace[dim]*Nint*ghost_precision;
    const void *src = ghost_spinor;
  
    int ghost_offset = (dir == QUDA_BACKWARDS) ? ghostOffset[dim][0] : ghostOffset[dim][1];
    void *ghost_dst = (char*)ghost_recv_buffer_d[bufferIndex] + ghost_precision*ghost_offset;

    if (ghost_precision == QUDA_HALF_PRECISION || ghost_precision == QUDA_QUARTER_PRECISION) len += nFace*ghostFace[dim]*sizeof(float);

    qudaMemcpyAsync(ghost_dst, src, len, hipMemcpyHostToDevice, *stream);
  }


  // pack the ghost zone into a contiguous buffer for communications
  void cudaColorSpinorField::packGhostExtended(const int nFace, const int R[], const QudaParity parity,
					       const int dim, const QudaDirection dir,
					       const int dagger, hipStream_t *stream, bool zero_copy)
  {
#ifdef MULTI_GPU
    int face_num = (dir == QUDA_BACKWARDS) ? 0 : (dir == QUDA_FORWARDS) ? 1 : 2;
    void *packBuffer[2*QUDA_MAX_DIM];
    MemoryLocation location[2*QUDA_MAX_DIM];

    if (zero_copy) {
      for (int d=0; d<4; d++) {
	packBuffer[2*d+0] = my_face_dim_dir_hd[bufferIndex][d][0];
	packBuffer[2*d+1] = my_face_dim_dir_hd[bufferIndex][d][1];
	location[2*d+0] = Host;
	location[2*d+1] = Host;
      }
    } else {
      for (int d=0; d<4; d++) {
	packBuffer[2*d+0] = my_face_dim_dir_d[bufferIndex][d][0];
	packBuffer[2*d+1] = my_face_dim_dir_d[bufferIndex][d][1];
	location[2*d+0] = Device;
	location[2*d+1] = Device;
      }
    }

    packFaceExtended(packBuffer, *this, location[0], nFace, R, dagger, parity, dim, face_num, *stream);
#else
    errorQuda("packGhostExtended not built on single-GPU build");
#endif

  }


  // copy data from host buffer into boundary region of device field
  void cudaColorSpinorField::unpackGhostExtended(const void* ghost_spinor, const int nFace, const QudaParity parity,
                                                 const int dim, const QudaDirection dir, 
                                                 const int dagger, hipStream_t* stream, bool zero_copy)
  {
    // First call the regular unpackGhost routine to copy data into the `usual' ghost-zone region 
    // of the data array 
    unpackGhost(ghost_spinor, nFace, dim, dir, dagger, stream);

    // Next step is to copy data from the ghost zone back to the interior region
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1); // (spin proj.) degrees of freedom

    int len = nFace*ghostFace[dim]*Nint;
    int offset = length + ghostOffset[dim][0];
    offset += (dir == QUDA_BACKWARDS) ? 0 : len;

#ifdef MULTI_GPU
    const int face_num = 2;
    const bool unpack = true;
    const int R[4] = {0,0,0,0};
    void *packBuffer[2*QUDA_MAX_DIM];
    MemoryLocation location[2*QUDA_MAX_DIM];

    if (zero_copy) {
      for (int d=0; d<4; d++) {
	packBuffer[2*d+0] = my_face_dim_dir_hd[bufferIndex][d][0];
	packBuffer[2*d+1] = my_face_dim_dir_hd[bufferIndex][d][1];
	location[2*d+0] = Host;
	location[2*d+1] = Host;
      }
    } else {
      for (int d=0; d<4; d++) {
	packBuffer[2*d+0] = my_face_dim_dir_d[bufferIndex][d][0];
	packBuffer[2*d+1] = my_face_dim_dir_d[bufferIndex][d][1];
	location[2*d+0] = Device;
	location[2*d+1] = Device;
      }
    }

    packFaceExtended(packBuffer, *this, location[0], nFace, R, dagger, parity, dim, face_num, *stream, unpack);
#else
    errorQuda("unpackGhostExtended not built on single-GPU build");
#endif
  }


  hipStream_t *stream;

  void cudaColorSpinorField::createComms(int nFace, bool spin_project) {

    allocateGhostBuffer(nFace,spin_project); // allocate the ghost buffer if not yet allocated

    // ascertain if this instance needs its comms buffers to be updated
    bool comms_reset = ghost_field_reset || // FIXME add send buffer check
      (my_face_h[0] != ghost_pinned_buffer_h[0]) || (my_face_h[1] != ghost_pinned_buffer_h[1]) || // pinned buffers
      (ghost_field_tex[0] != ghost_recv_buffer_d[0]) || (ghost_field_tex[1] != ghost_recv_buffer_d[1]) || // receive buffers
      ghost_precision_reset; // ghost_precision has changed

    if (!initComms || comms_reset) {

      LatticeField::createComms();

      // reinitialize the ghost receive pointers
      for (int i=0; i<nDimComms; ++i) {
	if (commDimPartitioned(i)) {
	  for (int b=0; b<2; b++) {
	    ghost[b][i] = static_cast<char*>(ghost_recv_buffer_d[b]) + ghostOffset[i][0]*ghost_precision;
	    if (ghost_precision == QUDA_HALF_PRECISION || ghost_precision == QUDA_QUARTER_PRECISION)
	      ghostNorm[b][i] = static_cast<char*>(ghost_recv_buffer_d[b]) + ghostNormOffset[i][0]*QUDA_SINGLE_PRECISION;
	  }
	}
      }

      ghost_precision_reset = false;
    }

    if (ghost_field_reset) destroyIPCComms();
    createIPCComms();
  }

  void cudaColorSpinorField::streamInit(hipStream_t *stream_p) {
    stream = stream_p;
  }

  void cudaColorSpinorField::pack(int nFace, int parity, int dagger, int stream_idx,
				  MemoryLocation location[2*QUDA_MAX_DIM], MemoryLocation location_label, double a, double b)
  {
    createComms(nFace); // must call this first

    const int dim=-1; // pack all partitioned dimensions
 
    packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[stream_idx], location, location_label, a, b);
  }

  void cudaColorSpinorField::packExtended(const int nFace, const int R[], const int parity, 
                                          const int dagger, const int dim,
                                          hipStream_t *stream_p, const bool zero_copy)
  {
    createComms(nFace); // must call this first

    stream = stream_p;
 
    packGhostExtended(nFace, R, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[zero_copy ? 0 : (Nstream-1)], zero_copy);
  }

  void cudaColorSpinorField::gather(int nFace, int dagger, int dir, hipStream_t* stream_p)
  {
    int dim = dir/2;

    // If stream_p != 0, use pack_stream, else use the stream array
    hipStream_t *pack_stream = (stream_p) ? stream_p : stream+dir;

    if (dir%2 == 0) {
      // backwards copy to host
      if (comm_peer2peer_enabled(0,dim)) return;

      sendGhost(my_face_dim_dir_h[bufferIndex][dim][0], nFace, dim, QUDA_BACKWARDS, dagger, pack_stream);
    } else {
      // forwards copy to host
      if (comm_peer2peer_enabled(1,dim)) return;

      sendGhost(my_face_dim_dir_h[bufferIndex][dim][1], nFace, dim, QUDA_FORWARDS, dagger, pack_stream);
    }
  }


  void cudaColorSpinorField::recvStart(int nFace, int d, int dagger, hipStream_t* stream_p, bool gdr) {

    // note this is scatter centric, so dir=0 (1) is send backwards
    // (forwards) and receive from forwards (backwards)

    int dim = d/2;
    int dir = d%2;
    if (!commDimPartitioned(dim)) return;
    if (gdr && !comm_gdr_enabled()) errorQuda("Requesting GDR comms but GDR is not enabled");

    if (dir == 0) { // receive from forwards
      // receive from the processor in the +1 direction
      if (comm_peer2peer_enabled(1,dim)) {
	comm_start(mh_recv_p2p_fwd[bufferIndex][dim]);
      } else if (gdr) {
        comm_start(mh_recv_rdma_fwd[bufferIndex][dim]);
      } else {
        comm_start(mh_recv_fwd[bufferIndex][dim]);
      }
    } else { // receive from backwards
      // receive from the processor in the -1 direction
      if (comm_peer2peer_enabled(0,dim)) {
	comm_start(mh_recv_p2p_back[bufferIndex][dim]);
      } else if (gdr) {
        comm_start(mh_recv_rdma_back[bufferIndex][dim]);
      } else {
        comm_start(mh_recv_back[bufferIndex][dim]);
      }
    }
  }


  void cudaColorSpinorField::sendStart(int nFace, int d, int dagger, hipStream_t* stream_p, bool gdr, bool remote_write) {

    // note this is scatter centric, so dir=0 (1) is send backwards
    // (forwards) and receive from forwards (backwards)

    int dim = d/2;
    int dir = d%2;
    if (!commDimPartitioned(dim)) return;
    if (gdr && !comm_gdr_enabled()) errorQuda("Requesting GDR comms but GDR is not enabled");

    int Nvec = (nSpin == 1 || ghost_precision == QUDA_DOUBLE_PRECISION) ? 2 : 4;
    int Nint = (nColor * nSpin * 2)/(nSpin == 4 ? 2 : 1); // (spin proj.) degrees of freedom
    int Npad = Nint/Nvec;

    if (!comm_peer2peer_enabled(dir,dim)) {
      if (dir == 0)
	if (gdr) comm_start(mh_send_rdma_back[bufferIndex][dim]);
	else comm_start(mh_send_back[bufferIndex][dim]);
      else
	if (gdr) comm_start(mh_send_rdma_fwd[bufferIndex][dim]);
	else comm_start(mh_send_fwd[bufferIndex][dim]);
    } else { // doing peer-to-peer
      hipStream_t *copy_stream = (stream_p) ? stream_p : stream + d;

      // if not using copy engine then the packing kernel will remotely write the halos
      if (!remote_write) {
        // all goes here
        void* ghost_dst = static_cast<char*>(ghost_remote_send_buffer_d[bufferIndex][dim][dir])
          + ghost_precision*ghostOffset[dim][(dir+1)%2];
        void *ghost_norm_dst = static_cast<char*>(ghost_remote_send_buffer_d[bufferIndex][dim][dir])
          + QUDA_SINGLE_PRECISION*ghostNormOffset[dim][(d+1)%2];

        if (ghost_precision != precision) pushKernelPackT(true);

        if (dim != 3 || getKernelPackT()) {

          void* ghost_dst = static_cast<char*>(ghost_remote_send_buffer_d[bufferIndex][dim][dir])
            + ghost_precision*ghostOffset[dim][(dir+1)%2];
          hipMemcpyAsync(ghost_dst,
                          my_face_dim_dir_d[bufferIndex][dim][dir],
                          ghost_face_bytes[dim],
                          hipMemcpyDeviceToDevice,
                          *copy_stream); // copy to forward processor

        } else if (this->TwistFlavor() != QUDA_TWIST_NONDEG_DOUBLET) {

          const int x4 = nDim==5 ? x[4] : 1;
          const int Nt_minus_offset = (volumeCB - nFace*ghostFace[3])/x4;

          int offset = 0;
          if (nSpin == 1) {
            offset = (dir == 0) ? 0 : Nt_minus_offset;
          } else if (nSpin == 4) {
            // !dagger: send lower components backwards, send upper components forwards
            // dagger: send upper components backwards, send lower components forwards
            bool upper = dagger ? true : false;
            if (dir == 1) upper = !upper;
            int lower_spin_offset = Npad*stride;
            if (dir == 0) {
              offset = upper ? 0 : lower_spin_offset;
            } else {
              offset = (upper) ? Nt_minus_offset : lower_spin_offset + Nt_minus_offset;
            }
          }

          size_t len = nFace*(ghostFace[3]/x4)*Nvec*ghost_precision;
          size_t dpitch = x4*len;
          size_t spitch = stride*Nvec*ghost_precision;

          for (int s=0; s<x4; s++) {
            void *dst = (char*)ghost_dst + s*len;
            void *src = (char*)v + (offset + s*(volumeCB/x4))*Nvec*ghost_precision;
            // start the copy
            hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToDevice, *copy_stream);

            if (ghost_precision == QUDA_HALF_PRECISION || ghost_precision == QUDA_QUARTER_PRECISION) {
              size_t len = nFace*(ghostFace[3]/x4)*sizeof(float);
              int norm_offset = (dir == 0) ? 0 : Nt_minus_offset*sizeof(float);
              void *dst = (char*)ghost_norm_dst + s*len;
              void *src = static_cast<char*>(norm) + norm_offset + s*(volumeCB/x4)*sizeof(float);
              hipMemcpyAsync(dst, src, len, hipMemcpyDeviceToDevice, *copy_stream);
            }
          }
        } else { // twisted doublet
          int flavorVolume = volume / 2;
          int flavorTFace  = ghostFace[3] / 2;
          int flavor1_Nt_minus1_offset = (flavorVolume - flavorTFace);
          int flavor2_Nt_minus1_offset = (volume - flavorTFace);
          int flavor1_offset = 0;
          int flavor2_offset = 0;
          // !dagger: send lower components backwards, send upper components forwards
          // dagger: send upper components backwards, send lower components forwards
          bool upper = dagger ? true : false; // Fwd is !Back
          if (dir == 1) upper = !upper;
          int lower_spin_offset = Npad*stride;//ndeg tm: stride=2*flavor_volume+pad
          if (upper) {
            flavor1_offset = (dir == 0 ? 0 : flavor1_Nt_minus1_offset);
            flavor2_offset = (dir == 0 ? flavorVolume : flavor2_Nt_minus1_offset);
          } else {
            flavor1_offset = lower_spin_offset + (dir == 0 ? 0 : flavor1_Nt_minus1_offset);
            flavor2_offset = lower_spin_offset + (dir == 0 ? flavorVolume : flavor2_Nt_minus1_offset);
          }

          // QUDA Memcpy NPad's worth.
          //  -- Dest will point to the right beginning PAD.
          //  -- Each Pad has size Nvec*Vsh Floats.
          //  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next

          void *src = static_cast<char*>(v) + flavor1_offset*Nvec*ghost_precision;
          size_t len = flavorTFace*Nvec*ghost_precision;
          size_t spitch = stride*Nvec*ghost_precision;//ndeg tm: stride=2*flavor_volume+pad
          size_t dpitch = 2*len;
          hipMemcpy2DAsync(ghost_dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToDevice, *copy_stream);

          src = static_cast<char*>(v) + flavor2_offset*Nvec*ghost_precision;
          hipMemcpy2DAsync(static_cast<char*>(ghost_dst)+len, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToDevice, *copy_stream);

          if (ghost_precision == QUDA_HALF_PRECISION || ghost_precision == QUDA_QUARTER_PRECISION) {
            int norm_offset = (dir == 0) ? 0 : flavor1_Nt_minus1_offset*sizeof(float);
            void *src = static_cast<char*>(norm) + norm_offset;
            size_t dpitch = flavorTFace*sizeof(float);
            size_t spitch = flavorVolume*sizeof(float);
            hipMemcpy2DAsync(ghost_norm_dst, dpitch, src, spitch, flavorTFace*sizeof(float), 2, hipMemcpyDeviceToDevice, *copy_stream);
          }
        }
      } // remote_write

      if (ghost_precision != precision) popKernelPackT();

      if (dir == 0) {
	// record the event
	qudaEventRecord(ipcCopyEvent[bufferIndex][0][dim], *copy_stream);
	// send to the processor in the -1 direction
	comm_start(mh_send_p2p_back[bufferIndex][dim]);
      } else {
	qudaEventRecord(ipcCopyEvent[bufferIndex][1][dim], *copy_stream);
	// send to the processor in the +1 direction
	comm_start(mh_send_p2p_fwd[bufferIndex][dim]);
      }
    }
  }

  void cudaColorSpinorField::commsStart(int nFace, int dir, int dagger, hipStream_t* stream_p, bool gdr_send, bool gdr_recv) {
    recvStart(nFace, dir, dagger, stream_p, gdr_recv);
    sendStart(nFace, dir, dagger, stream_p, gdr_send);
  }


  static bool complete_recv_fwd[QUDA_MAX_DIM] = { };
  static bool complete_recv_back[QUDA_MAX_DIM] = { };
  static bool complete_send_fwd[QUDA_MAX_DIM] = { };
  static bool complete_send_back[QUDA_MAX_DIM] = { };

  int cudaColorSpinorField::commsQuery(int nFace, int d, int dagger, hipStream_t *stream_p, bool gdr_send, bool gdr_recv) {

    // note this is scatter centric, so dir=0 (1) is send backwards
    // (forwards) and receive from forwards (backwards)

    int dim = d/2;
    int dir = d%2;

    if (!commDimPartitioned(dim)) return 1;
    if ((gdr_send || gdr_recv) && !comm_gdr_enabled()) errorQuda("Requesting GDR comms but GDR is not enabled");

    if (dir==0) {

      // first query send to backwards
      if (comm_peer2peer_enabled(0,dim)) {
	if (!complete_send_back[dim]) complete_send_back[dim] = comm_query(mh_send_p2p_back[bufferIndex][dim]);
      } else if (gdr_send) {
	if (!complete_send_back[dim]) complete_send_back[dim] = comm_query(mh_send_rdma_back[bufferIndex][dim]);
      } else {
	if (!complete_send_back[dim]) complete_send_back[dim] = comm_query(mh_send_back[bufferIndex][dim]);
      }

      // second query receive from forwards
      if (comm_peer2peer_enabled(1,dim)) {
	if (!complete_recv_fwd[dim]) complete_recv_fwd[dim] = comm_query(mh_recv_p2p_fwd[bufferIndex][dim]);
      } else if (gdr_recv) {
	if (!complete_recv_fwd[dim]) complete_recv_fwd[dim] = comm_query(mh_recv_rdma_fwd[bufferIndex][dim]);
      } else {
	if (!complete_recv_fwd[dim]) complete_recv_fwd[dim] = comm_query(mh_recv_fwd[bufferIndex][dim]);
      }

      if (complete_recv_fwd[dim] && complete_send_back[dim]) {
	complete_send_back[dim] = false;
	complete_recv_fwd[dim] = false;
	return 1;
      }

    } else { // dir == 1

      // first query send to forwards
      if (comm_peer2peer_enabled(1,dim)) {
	if (!complete_send_fwd[dim]) complete_send_fwd[dim] = comm_query(mh_send_p2p_fwd[bufferIndex][dim]);
      } else if (gdr_send) {
	if (!complete_send_fwd[dim]) complete_send_fwd[dim] = comm_query(mh_send_rdma_fwd[bufferIndex][dim]);
      } else {
	if (!complete_send_fwd[dim]) complete_send_fwd[dim] = comm_query(mh_send_fwd[bufferIndex][dim]);
      }

      // second query receive from backwards
      if (comm_peer2peer_enabled(0,dim)) {
	if (!complete_recv_back[dim]) complete_recv_back[dim] = comm_query(mh_recv_p2p_back[bufferIndex][dim]);
      } else if (gdr_recv) {
	if (!complete_recv_back[dim]) complete_recv_back[dim] = comm_query(mh_recv_rdma_back[bufferIndex][dim]);
      } else {
	if (!complete_recv_back[dim]) complete_recv_back[dim] = comm_query(mh_recv_back[bufferIndex][dim]);
      }

      if (complete_recv_back[dim] && complete_send_fwd[dim]) {
	complete_send_fwd[dim] = false;
	complete_recv_back[dim] = false;
	return 1;
      }

    }

    return 0;
  }

  void cudaColorSpinorField::commsWait(int nFace, int d, int dagger, hipStream_t *stream_p, bool gdr_send, bool gdr_recv) {

    // note this is scatter centric, so dir=0 (1) is send backwards
    // (forwards) and receive from forwards (backwards)

    int dim = d/2;
    int dir = d%2;

    if (!commDimPartitioned(dim)) return;
    if ( (gdr_send && gdr_recv) && !comm_gdr_enabled()) errorQuda("Requesting GDR comms but GDR is not enabled");

    if (dir==0) {

      // first wait on send to backwards
      if (comm_peer2peer_enabled(0,dim)) {
	comm_wait(mh_send_p2p_back[bufferIndex][dim]);
	hipEventSynchronize(ipcCopyEvent[bufferIndex][0][dim]);
      } else if (gdr_send) {
	comm_wait(mh_send_rdma_back[bufferIndex][dim]);
      } else {
	comm_wait(mh_send_back[bufferIndex][dim]);
      }

      // second wait on receive from forwards
      if (comm_peer2peer_enabled(1,dim)) {
	comm_wait(mh_recv_p2p_fwd[bufferIndex][dim]);
	hipEventSynchronize(ipcRemoteCopyEvent[bufferIndex][1][dim]);
      } else if (gdr_recv) {
	comm_wait(mh_recv_rdma_fwd[bufferIndex][dim]);
      } else {
	comm_wait(mh_recv_fwd[bufferIndex][dim]);
      }

    } else {

      // first wait on send to forwards
      if (comm_peer2peer_enabled(1,dim)) {
	comm_wait(mh_send_p2p_fwd[bufferIndex][dim]);
	hipEventSynchronize(ipcCopyEvent[bufferIndex][1][dim]);
      } else if (gdr_send) {
	comm_wait(mh_send_rdma_fwd[bufferIndex][dim]);
      } else {
	comm_wait(mh_send_fwd[bufferIndex][dim]);
      }

      // second wait on receive from backwards
      if (comm_peer2peer_enabled(0,dim)) {
	comm_wait(mh_recv_p2p_back[bufferIndex][dim]);
	hipEventSynchronize(ipcRemoteCopyEvent[bufferIndex][0][dim]);
      } else if (gdr_recv) {
	comm_wait(mh_recv_rdma_back[bufferIndex][dim]);
      } else {
	comm_wait(mh_recv_back[bufferIndex][dim]);
      }

    }

    return;
  }

  void cudaColorSpinorField::scatter(int nFace, int dagger, int dim_dir, hipStream_t* stream_p)
  {
    // note this is scatter centric, so input expects dir=0 (1) is send backwards
    // (forwards) and receive from forwards (backwards), so here we need flip to receive centric

    int dim = dim_dir/2;
    int dir = (dim_dir+1)%2; // dir = 1 - receive from forwards, dir == 0 recive from backwards
    if (!commDimPartitioned(dim)) return;

    if (comm_peer2peer_enabled(dir,dim)) return;
    unpackGhost(from_face_dim_dir_h[bufferIndex][dim][dir], nFace, dim, dir == 0 ? QUDA_BACKWARDS : QUDA_FORWARDS, dagger, stream_p);
  }

  void cudaColorSpinorField::scatter(int nFace, int dagger, int dim_dir)
  {
    // note this is scatter centric, so dir=0 (1) is send backwards
    // (forwards) and receive from forwards (backwards), so here we need flip to receive centric

    int dim = dim_dir/2;
    int dir = (dim_dir+1)%2; // dir = 1 - receive from forwards, dir == 0 receive from backwards
    if (!commDimPartitioned(dim)) return;

    if (comm_peer2peer_enabled(dir,dim)) return;
    unpackGhost(from_face_dim_dir_h[bufferIndex][dim][dir], nFace, dim, dir == 0 ? QUDA_BACKWARDS : QUDA_FORWARDS, dagger, &stream[dim_dir]);
  }

  void cudaColorSpinorField::scatterExtended(int nFace, int parity, int dagger, int dim_dir)
  {
    bool zero_copy = false;
    int dim = dim_dir/2;
    int dir = (dim_dir+1)%2; // dir = 1 - receive from forwards, dir == 0 receive from backwards
    if (!commDimPartitioned(dim)) return;
    unpackGhostExtended(from_face_dim_dir_h[bufferIndex][dim][dir], nFace, static_cast<QudaParity>(parity), dim, dir == 0 ? QUDA_BACKWARDS : QUDA_FORWARDS, dagger, &stream[2*dim/*+0*/], zero_copy);
  }
 
  void cudaColorSpinorField::exchangeGhost(QudaParity parity, int nFace, int dagger, const MemoryLocation *pack_destination_,
					   const MemoryLocation *halo_location_, bool gdr_send, bool gdr_recv,
					   QudaPrecision ghost_precision_)  const {

    if (ghost_precision_ != QUDA_INVALID_PRECISION && ghost_precision != ghost_precision_) {
      ghost_precision_reset = true;
      ghost_precision_old = ghost_precision;
      ghost_precision = ghost_precision_;
    }

    // not overriding the ghost precision, but we did previously so need to update
    if (ghost_precision == QUDA_INVALID_PRECISION && ghost_precision != precision) {
      ghost_precision_reset = true;
      ghost_precision_old = ghost_precision;
      ghost_precision = precision;
    }

    if ((gdr_send || gdr_recv) && !comm_gdr_enabled()) errorQuda("Requesting GDR comms but GDR is not enabled");
    pushKernelPackT(true); // ensure kernel packing is enabled for all dimensions
    const_cast<cudaColorSpinorField&>(*this).streamInit(streams); // ensures streams are set (needed for p2p)
    const_cast<cudaColorSpinorField&>(*this).createComms(nFace, false);

    // first set default values to device if needed
    MemoryLocation pack_destination[2*QUDA_MAX_DIM], halo_location[2*QUDA_MAX_DIM];
    for (int i=0; i<2*nDimComms; i++) {
      pack_destination[i] = pack_destination_ ? pack_destination_[i] : Device;
      halo_location[i] = halo_location_ ? halo_location_[i] : Device;
    }

    // Contiguous send buffers and we aggregate copies to reduce
    // latency.  Only if all locations are "Device" and no p2p
    bool fused_pack_memcpy = true;

    // Contiguous recv buffers and we aggregate copies to reduce
    // latency.  Only if all locations are "Device" and no p2p
    bool fused_halo_memcpy = true;

    bool pack_host = false; // set to true if any of the ghost packing is being done to Host memory
    bool halo_host = false; // set to true if the final halos will be left in Host memory

    void *send[2*QUDA_MAX_DIM];
    for (int d=0; d<nDimComms; d++) {
      for (int dir=0; dir<2; dir++) {
	send[2*d+dir] = pack_destination[2*d+dir] == Host ? my_face_dim_dir_hd[bufferIndex][d][dir] : my_face_dim_dir_d[bufferIndex][d][dir];
	ghost_buf[2*d+dir] = halo_location[2*d+dir] == Host ? from_face_dim_dir_hd[bufferIndex][d][dir] : from_face_dim_dir_d[bufferIndex][d][dir];
      }

      // if doing p2p, then we must pack to and load the halo from device memory
      for (int dir=0; dir<2; dir++) {
	if (comm_peer2peer_enabled(dir,d)) { pack_destination[2*d+dir] = Device; halo_location[2*d+1-dir] = Device; }
      }

      // if zero-copy packing or p2p is enabled then we cannot do fused memcpy
      if (pack_destination[2*d+0] != Device || pack_destination[2*d+1] != Device || comm_peer2peer_enabled_global()) fused_pack_memcpy = false;
      // if zero-copy halo read or p2p is enabled then we cannot do fused memcpy
      if (halo_location[2*d+0] != Device || halo_location[2*d+1] != Device || comm_peer2peer_enabled_global()) fused_halo_memcpy = false;

      if (pack_destination[2*d+0] == Host || pack_destination[2*d+1] == Host) pack_host = true;
      if (halo_location[2*d+0] == Host || halo_location[2*d+1] == Host) halo_host = true;
    }

    // Error if zero-copy and p2p for now
    if ( (pack_host || halo_host) && comm_peer2peer_enabled_global()) errorQuda("Cannot use zero-copy memory with peer-to-peer comms yet");

    genericPackGhost(send, *this, parity, nFace, dagger, pack_destination); // FIXME - need support for asymmetric topologies

    size_t total_bytes = 0;
    for (int i=0; i<nDimComms; i++) if (comm_dim_partitioned(i)) total_bytes += 2*ghost_face_bytes[i]; // 2 for fwd/bwd

    if (!gdr_send)  {
      if (!fused_pack_memcpy) {
	for (int i=0; i<nDimComms; i++) {
	  if (comm_dim_partitioned(i)) {
	    if (pack_destination[2*i+0] == Device && !comm_peer2peer_enabled(0,i) && // fuse forwards and backwards if possible
		pack_destination[2*i+1] == Device && !comm_peer2peer_enabled(1,i)) {
	      hipMemcpyAsync(my_face_dim_dir_h[bufferIndex][i][0], my_face_dim_dir_d[bufferIndex][i][0], 2*ghost_face_bytes[i], hipMemcpyDeviceToHost, 0);
	    } else {
	      if (pack_destination[2*i+0] == Device && !comm_peer2peer_enabled(0,i))
		hipMemcpyAsync(my_face_dim_dir_h[bufferIndex][i][0], my_face_dim_dir_d[bufferIndex][i][0], ghost_face_bytes[i], hipMemcpyDeviceToHost, 0);
	      if (pack_destination[2*i+1] == Device && !comm_peer2peer_enabled(1,i))
		hipMemcpyAsync(my_face_dim_dir_h[bufferIndex][i][1], my_face_dim_dir_d[bufferIndex][i][1], ghost_face_bytes[i], hipMemcpyDeviceToHost, 0);
	    }
	  }
	}
      } else if (total_bytes && !pack_host) {
	hipMemcpyAsync(my_face_h[bufferIndex], ghost_send_buffer_d[bufferIndex], total_bytes, hipMemcpyDeviceToHost, 0);
      }
    }

    // prepost receive
    for (int i=0; i<2*nDimComms; i++) const_cast<cudaColorSpinorField*>(this)->recvStart(nFace, i, dagger, 0, gdr_recv);

    bool sync = pack_host ? true : false; // no p2p if pack_host so we need to synchronize
    // if not p2p in any direction then need to synchronize before MPI
    for (int i=0; i<nDimComms; i++) if (!comm_peer2peer_enabled(0,i) || !comm_peer2peer_enabled(1,i)) sync = true;
    if (sync) qudaDeviceSynchronize(); // need to make sure packing and/or memcpy has finished before kicking off MPI

    for (int p2p=0; p2p<2; p2p++) {
      for (int dim=0; dim<nDimComms; dim++) {
	for (int dir=0; dir<2; dir++) {
	  if ( (comm_peer2peer_enabled(dir,dim) + p2p) % 2 == 0 ) { // issue non-p2p transfers first
	    const_cast<cudaColorSpinorField*>(this)->sendStart(nFace, 2*dim+dir, dagger, 0, gdr_send);
	  }
	}
      }
    }

    bool comms_complete[2*QUDA_MAX_DIM] = { };
    int comms_done = 0;
    while (comms_done < 2*nDimComms) { // non-blocking query of each exchange and exit once all have completed
      for (int dim=0; dim<nDimComms; dim++) {
	for (int dir=0; dir<2; dir++) {
	  if (!comms_complete[dim*2+dir]) {
	    comms_complete[2*dim+dir] = const_cast<cudaColorSpinorField*>(this)->commsQuery(nFace, 2*dim+dir, dagger, 0, gdr_send, gdr_recv);
	    if (comms_complete[2*dim+dir]) {
	      comms_done++;
	      if (comm_peer2peer_enabled(1-dir,dim)) qudaStreamWaitEvent(0, ipcRemoteCopyEvent[bufferIndex][1-dir][dim], 0);
	    }
	  }
	}
      }
    }

    if (!gdr_recv) {
      if (!fused_halo_memcpy) {
	for (int i=0; i<nDimComms; i++) {
	  if (comm_dim_partitioned(i)) {
	    if (halo_location[2*i+0] == Device && !comm_peer2peer_enabled(0,i) && // fuse forwards and backwards if possible
		halo_location[2*i+1] == Device && !comm_peer2peer_enabled(1,i)) {
	      hipMemcpyAsync(from_face_dim_dir_d[bufferIndex][i][0], from_face_dim_dir_h[bufferIndex][i][0], 2*ghost_face_bytes[i], hipMemcpyHostToDevice, 0);
	    } else {
	      if (halo_location[2*i+0] == Device && !comm_peer2peer_enabled(0,i))
		hipMemcpyAsync(from_face_dim_dir_d[bufferIndex][i][0], from_face_dim_dir_h[bufferIndex][i][0], ghost_face_bytes[i], hipMemcpyHostToDevice, 0);
	      if (halo_location[2*i+1] == Device && !comm_peer2peer_enabled(1,i))
		hipMemcpyAsync(from_face_dim_dir_d[bufferIndex][i][1], from_face_dim_dir_h[bufferIndex][i][1], ghost_face_bytes[i], hipMemcpyHostToDevice, 0);
	    }
	  }
	}
      } else if (total_bytes && !halo_host) {
	hipMemcpyAsync(ghost_recv_buffer_d[bufferIndex], from_face_h[bufferIndex], total_bytes, hipMemcpyHostToDevice, 0);
      }
    }

    popKernelPackT(); // restore kernel packing
  }

  std::ostream& operator<<(std::ostream &out, const cudaColorSpinorField &a) {
    out << (const ColorSpinorField&)a;
    out << "v = " << a.v << std::endl;
    out << "norm = " << a.norm << std::endl;
    out << "alloc = " << a.alloc << std::endl;
    out << "init = " << a.init << std::endl;
    return out;
  }

//! for composite fields:
  cudaColorSpinorField& cudaColorSpinorField::Component(const int idx) const {
    
    if (this->IsComposite()) {
      if (idx < this->CompositeDim()) {//setup eigenvector form the set
        return *(dynamic_cast<cudaColorSpinorField*>(components[idx])); 
      }
      else{
        errorQuda("Incorrect component index...");
      }
    }
    errorQuda("Cannot get requested component");
    exit(-1);
  }

//copyCuda currently cannot not work with set of spinor fields..
  void cudaColorSpinorField::CopySubset(cudaColorSpinorField &dst, const int range, const int first_element) const{
#if 0
    if (first_element < 0) errorQuda("\nError: trying to set negative first element.\n");
    if (siteSubset == QUDA_PARITY_SITE_SUBSET && this->EigvId() == -1) {
      if (first_element == 0 && range == this->EigvDim())
      {
        if (range != dst.EigvDim())errorQuda("\nError: eigenvector range to big.\n");
        checkField(dst, *this);
        copyCuda(dst, *this);
      }
      else if ((first_element+range) < this->EigvDim()) 
      {//setup eigenvector subset

        cudaColorSpinorField *eigv_subset;

        ColorSpinorParam param;

        param.nColor = nColor;
        param.nSpin = nSpin;
        param.twistFlavor = twistFlavor;
        param.precision = precision;
        param.nDim = nDim;
        param.pad = pad;
        param.siteSubset = siteSubset;
        param.siteOrder = siteOrder;
        param.fieldOrder = fieldOrder;
        param.gammaBasis = gammaBasis;
        memcpy(param.x, x, nDim*sizeof(int));
        param.create = QUDA_REFERENCE_FIELD_CREATE;
 
        param.eigv_dim  = range;
        param.eigv_id   = -1;
        param.v = (void*)((char*)v + first_element*eigv_bytes);
        param.norm = (void*)((char*)norm + first_element*eigv_norm_bytes);

        eigv_subset = new cudaColorSpinorField(param);

        //Not really needed:
        eigv_subset->eigenvectors.reserve(param.eigv_dim);
        for (int id = first_element; id < (first_element+range); id++)
        {
            param.eigv_id = id;
            eigv_subset->eigenvectors.push_back(new cudaColorSpinorField(*this, param));
        }
        checkField(dst, *eigv_subset);
        copyCuda(dst, *eigv_subset);

        delete eigv_subset;
      } else {
        errorQuda("Incorrect eigenvector dimension...");
      }
    } else{
      errorQuda("Eigenvector must be a parity spinor");
      exit(-1);
    }
#endif
  }

  void cudaColorSpinorField::getTexObjectInfo() const
  {
#ifdef USE_TEXTURE_OBJECTS
    printfQuda("\nPrint texture info for the field:\n");
    std::cout << *this;
    hipResourceDesc resDesc;
    //memset(&resDesc, 0, sizeof(resDesc));
    hipGetTextureObjectResourceDesc(&resDesc, this->Tex());
    printfQuda("\nDevice pointer: %p\n", resDesc.res.linear.devPtr);
    printfQuda("\nVolume (in bytes): %lu\n", resDesc.res.linear.sizeInBytes);
    if (resDesc.resType == hipResourceTypeLinear) printfQuda("\nResource type: linear \n");
#endif
  }

  void cudaColorSpinorField::Source(const QudaSourceType sourceType, const int st, const int s, const int c) {
    ColorSpinorParam param(*this);
    param.fieldOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;
    param.location = QUDA_CPU_FIELD_LOCATION;
    param.create = (sourceType == QUDA_POINT_SOURCE ? QUDA_ZERO_FIELD_CREATE : QUDA_NULL_FIELD_CREATE);

    cpuColorSpinorField tmp(param);
    tmp.Source(sourceType, st, s, c);
    *this = tmp;
  }


  // This existed before but was never called. It kept giving me segfaults,
  // but it would be ideal if I could've gotten it to work.
  /*
  template<typename Float,int Ns, int Nc>
  void genericCudaPrintVector(const cudaColorSpinorField& field, unsigned int i)
  {
    typedef typename colorspinor_mapper<Float,Ns,Nc>::type ColorSpinor;
    ColorSpinor csmap(field);
    typedef typename mapper<Float>::type TrueFloat;
    TrueFloat* data_gpu=0;
    hipMalloc((void**)&data_gpu, 2*Ns*Nc*sizeof(TrueFloat));
    csmap.load(data_gpu,i);
    TrueFloat* data_cpu = new TrueFloat[2*Ns*Nc];
    hipMemcpy(data_cpu, data_gpu, 2*Ns*Nc*sizeof(TrueFloat), hipMemcpyDeviceToHost);
    hipFree(data_gpu);
    for (int s = 0; s < Ns; s++) {
      printfQuda("x = %u, s = %d, { ", i, s);
      for (int c = 0; c < Nc; c++) {
        printfQuda("(%f,%f)", data_cpu[(s*Nc+c)*2], data_cpu[(s*Nc+c)*2+1]);
        printfQuda("%s", c == Nc-1 ? " " : " , ");
      }
      printfQuda("}\n");
    }
    delete[] data_cpu;
  }*/

  template <typename StoreType, int Ns, int Nc, QudaFieldOrder FieldOrder>
  void genericCudaPrintVector(const cudaColorSpinorField& field, unsigned int i)
  {

    typedef colorspinor::AccessorCB<StoreType,Ns,Nc,1,FieldOrder> AccessorType;

    AccessorType A(field);

    // Register type
    typedef typename scalar<typename mapper<StoreType>::type>::type Float;

    // Allocate a real+imag component for the storage type.
    StoreType indiv_num[2];

    // Allocate space for the full site.
    Float* data_cpu = new Float[2*Ns*Nc];

    // Grab the pointer to the field.
    complex<StoreType>* field_ptr = (complex<StoreType>*)field.V();

    // Grab the pointer to the norm field. Might be ignored as appropriate.
    float* norm_ptr = (float*)field.Norm();
    float scale = 1.0;

    if (isFixed<StoreType>::value)
    {
      hipMemcpy(&scale, &norm_ptr[i], sizeof(float), hipMemcpyDeviceToHost);
      scale *= fixedInvMaxValue<StoreType>::value;
    }

    for (int s = 0; s < Ns; s++)
    {
      for (int c = 0; c < Nc; c++)
      {
        hipMemcpy(indiv_num, &field_ptr[A.index(i%2,i/2,s,c,0)], 2*sizeof(StoreType), hipMemcpyDeviceToHost);
        data_cpu[2*(c+Nc*s)] = scale*static_cast<Float>(indiv_num[0]);
        data_cpu[2*(c+Nc*s)+1] = scale*static_cast<Float>(indiv_num[1]);
      }
    }
    // print
    for (int s = 0; s < Ns; s++) {
      printfQuda("x = %u, s = %d, { ", i, s);
      for (int c = 0; c < Nc; c++) {
        printfQuda("(%f,%f)", data_cpu[(s*Nc+c)*2], data_cpu[(s*Nc+c)*2+1]);
        printfQuda("%s", c == Nc-1 ? " " : " , ");
      }
      printfQuda("}\n");
    }
    //for (int j = 0; j < 2*Ns*Nc; j++)
    //  printfQuda("%f\n", data_cpu[j]);
    
    delete[] data_cpu;

  }

  template<typename Float,int Ns, int Nc>
  void genericCudaPrintVector(const cudaColorSpinorField& field, unsigned int i)
  {
    switch (field.FieldOrder()) {
    case QUDA_FLOAT_FIELD_ORDER:
      genericCudaPrintVector<Float,Ns,Nc,QUDA_FLOAT_FIELD_ORDER>(field,i);
      break;
    case QUDA_FLOAT2_FIELD_ORDER:
      genericCudaPrintVector<Float,Ns,Nc,QUDA_FLOAT2_FIELD_ORDER>(field,i);
      break;
    case QUDA_FLOAT4_FIELD_ORDER:
      genericCudaPrintVector<Float,Ns,Nc,QUDA_FLOAT4_FIELD_ORDER>(field,i);
      break;
    case QUDA_SPACE_SPIN_COLOR_FIELD_ORDER:
      genericCudaPrintVector<Float,Ns,Nc,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>(field,i);
      break;
    case QUDA_SPACE_COLOR_SPIN_FIELD_ORDER:
      genericCudaPrintVector<Float,Ns,Nc,QUDA_SPACE_COLOR_SPIN_FIELD_ORDER>(field,i);
      break;
    default:
      errorQuda("Unsupported field order %d", field.FieldOrder());
    }
  }

  template<typename Float>
  void genericCudaPrintVector(const cudaColorSpinorField& field, unsigned int i) {
    if (field.Ncolor() == 3 && field.Nspin() == 4)  {
      genericCudaPrintVector<Float,4,3>(field,i);
    }
    else if (field.Ncolor() == 3 && field.Nspin() == 1)  {
      genericCudaPrintVector<Float,1,3>(field,i);
    }
    else if (field.Ncolor() == 6 && field.Nspin() == 2) { // wilson free field MG
      genericCudaPrintVector<Float,2,6>(field,i);
    }
    else if (field.Ncolor() == 24 && field.Nspin() == 2) { // common value for Wilson, also staggered free field
      genericCudaPrintVector<Float,2,24>(field,i);
    }
    else if (field.Ncolor() == 32 && field.Nspin() == 2) {
      genericCudaPrintVector<Float,2,32>(field,i);
    } else {
      errorQuda("Not supported Ncolor = %d, Nspin = %d", field.Ncolor(), field.Nspin());
    }
  }

  void cudaColorSpinorField::PrintVector(unsigned int i) {
    
    switch (this->precision)
    {
      case QUDA_QUARTER_PRECISION:
        genericCudaPrintVector<char>(*this,i);
        break;
      case QUDA_HALF_PRECISION:
        genericCudaPrintVector<short>(*this,i);
        break;
      case QUDA_SINGLE_PRECISION:
        genericCudaPrintVector<float>(*this,i);
        break;
      case QUDA_DOUBLE_PRECISION:
        genericCudaPrintVector<double>(*this, i);
        break;
      default:
        errorQuda("Unsupported precision = %d\n", this->precision);
    }

    /*
    ColorSpinorParam param(*this);
    param.fieldOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;
    param.location = QUDA_CPU_FIELD_LOCATION;
    param.create = QUDA_NULL_FIELD_CREATE;

    cpuColorSpinorField tmp(param);
    tmp = *this;
    tmp.PrintVector(i);*/
  }


} // namespace quda
