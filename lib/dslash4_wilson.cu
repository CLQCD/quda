#ifndef USE_LEGACY_DSLASH

#include <gauge_field.h>
#include <color_spinor_field.h>
#include <dslash.h>
#include <worker.h>

#include <dslash_policy.cuh>
#include <kernels/dslash_wilson.cuh>

/**
   This is the basic gauged Wilson operator

   TODO
   - gauge fix support
   - ghost texture support in accessors
   - CPU support
*/

namespace quda {

  /**
     @brief This is a helper class that is used to instantiate the
     correct templated kernel for the dslash.
   */
  template <typename Float, int nDim, int nColor, int nParity, bool dagger, bool xpay, KernelType kernel_type, typename Arg>
  struct WilsonLaunch {
    static constexpr const char *kernel = "quda::wilsonGPU"; // kernel name for jit compilation
    template <typename Dslash>
    inline static void launch(Dslash &dslash, TuneParam &tp, Arg &arg, const hipStream_t &stream) {
      dslash.launch(wilsonGPU<Float,nDim,nColor,nParity,dagger,xpay,kernel_type,Arg>, tp, arg, stream);
    }
  };

  template <typename Float, int nDim, int nColor, typename Arg>
  class Wilson : public Dslash<Float> {

  protected:
    Arg &arg;
    const ColorSpinorField &in;

  public:

    Wilson(Arg &arg, const ColorSpinorField &out, const ColorSpinorField &in)
      : Dslash<Float>(arg, out, in, "kernels/dslash_wilson.cuh"), arg(arg), in(in) { }

    virtual ~Wilson() { }

    void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      Dslash<Float>::setParam(arg);

      if (arg.pack_threads && arg.kernel_type == INTERIOR_KERNEL) {
        arg.blocks_per_dir = tp.aux.x;
        arg.setPack(true); // need to recompute for updated block_per_dir
        arg.in.resetGhost(in, this->packBuffer);
        tp.grid.x += arg.pack_blocks;
      }

      Dslash<Float>::template instantiate<WilsonLaunch,nDim,nColor>(tp, arg, stream);
    }

    TuneKey tuneKey() const {
      if (arg.pack_blocks > 0 && arg.kernel_type == INTERIOR_KERNEL) {
        return TuneKey(in.VolString(), typeid(*this).name(),
                       Dslash<Float>::aux_pack);
      } else {
        return TuneKey(in.VolString(), typeid(*this).name(),
                       Dslash<Float>::aux[arg.kernel_type]);
      }
    }
  };

  template <typename Float, int nColor, QudaReconstructType recon>
  void ApplyWilson(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
                   double kappa, const ColorSpinorField &x, int parity, bool dagger,
                   const int *comm_override, TimeProfile &profile)
  {
    constexpr int nDim = 4;
    WilsonArg<Float,nColor,recon> arg(out, in, U, kappa, x, parity, dagger, comm_override);
    Wilson<Float,nDim,nColor,WilsonArg<Float,nColor,recon> > wilson(arg, out, in);

    dslash::DslashPolicyTune<decltype(wilson)> policy(wilson, const_cast<cudaColorSpinorField*>(static_cast<const cudaColorSpinorField*>(&in)), in.VolumeCB(), in.GhostFaceCB(), profile);
    policy.apply(0);

    checkCudaError();
  }

  // template on the gauge reconstruction
  template <typename Float, int nColor>
  void ApplyWilson(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
                   double kappa, const ColorSpinorField &x, int parity, bool dagger,
                   const int *comm_override, TimeProfile &profile)
  {
    if (U.Reconstruct()== QUDA_RECONSTRUCT_NO) {
      ApplyWilson<Float,nColor,QUDA_RECONSTRUCT_NO>(out, in, U, kappa, x, parity, dagger, comm_override, profile);
    } else if (U.Reconstruct()== QUDA_RECONSTRUCT_12) {
      ApplyWilson<Float,nColor,QUDA_RECONSTRUCT_12>(out, in, U, kappa, x, parity, dagger, comm_override, profile);
    } else if (U.Reconstruct()== QUDA_RECONSTRUCT_8) {
      ApplyWilson<Float,nColor,QUDA_RECONSTRUCT_8>(out, in, U, kappa, x, parity, dagger, comm_override, profile);
    } else {
      errorQuda("Unsupported reconstruct type %d\n", U.Reconstruct());
    }
  }

  // template on the number of colors
  template <typename Float>
  void ApplyWilson(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
                   double kappa, const ColorSpinorField &x, int parity, bool dagger,
                   const int *comm_override, TimeProfile &profile)
  {
    if (in.Ncolor() == 3) {
      ApplyWilson<Float,3>(out, in, U, kappa, x, parity, dagger, comm_override, profile);
    } else {
      errorQuda("Unsupported number of colors %d\n", U.Ncolor());
    }
  }

  //Apply the Wilson operator
  //out(x) = M*in = - kappa*\sum_mu U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu)
  //Uses the kappa normalization for the Wilson operator.
  void ApplyWilson(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
                   double kappa, const ColorSpinorField &x, int parity, bool dagger,
                   const int *comm_override, TimeProfile &profile)
  {
#ifdef GPU_WILSON_DIRAC
    if (in.V() == out.V()) errorQuda("Aliasing pointers");
    if (in.FieldOrder() != out.FieldOrder())
      errorQuda("Field order mismatch in = %d, out = %d", in.FieldOrder(), out.FieldOrder());
    
    // check all precisions match
    checkPrecision(out, in, U);

    // check all locations match
    checkLocation(out, in, U);

    if (U.Precision() == QUDA_DOUBLE_PRECISION) {
      ApplyWilson<double>(out, in, U, kappa, x, parity, dagger, comm_override, profile);
    } else if (U.Precision() == QUDA_SINGLE_PRECISION) {
      ApplyWilson<float>(out, in, U, kappa, x, parity, dagger, comm_override, profile);
    } else if (U.Precision() == QUDA_HALF_PRECISION) {
      ApplyWilson<short>(out, in, U, kappa, x, parity, dagger, comm_override, profile);
    } else if (U.Precision() == QUDA_QUARTER_PRECISION) {
      ApplyWilson<char>(out, in, U, kappa, x, parity, dagger, comm_override, profile);
    } else {
      errorQuda("Unsupported precision %d\n", U.Precision());
    }
#else
    errorQuda("Wilson dslash has not been built");
#endif // GPU_WILSON_DIRAC
  }


} // namespace quda

#endif
