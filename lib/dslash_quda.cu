#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>

#include <color_spinor_field.h>
#include <clover_field.h>

// these control the Wilson-type actions
#ifdef GPU_WILSON_DIRAC
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_INTER
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR
//#define DIRECT_ACCESS_CLOVER
#endif // GPU_WILSON_DIRAC

//these are access control for staggered action
#ifdef GPU_STAGGERED_DIRAC
#if (__COMPUTE_CAPABILITY__ >= 300) // Kepler works best with texture loads only
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
//#define DIRECT_ACCESS_SPINOR
//#define DIRECT_ACCESS_ACCUM
//#define DIRECT_ACCESS_INTER
//#define DIRECT_ACCESS_PACK
#else // fermi
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
#define DIRECT_ACCESS_SPINOR
//#define DIRECT_ACCESS_ACCUM
//#define DIRECT_ACCESS_INTER
//#define DIRECT_ACCESS_PACK
#endif
#endif // GPU_STAGGERED_DIRAC

#include <quda_internal.h>
#include <dslash_quda.h>
#include <sys/time.h>
#include <blas_quda.h>
#include <face_quda.h>

#include <inline_ptx.h>

namespace quda {

  namespace dslash_aux {
#include <dslash_constants.h>
#include <dslash_textures.h>
#include <dslash_index.cuh>
#include <io_spinor.h>

#include <tm_core.h>              // solo twisted mass kernel
#include <tmc_core.h>              // solo twisted mass kernel
#include <clover_def.h>           // kernels for applying the clover term alone
  }

#ifndef DSLASH_SHARED_FLOATS_PER_THREAD
#define DSLASH_SHARED_FLOATS_PER_THREAD 0
#endif

#ifndef CLOVER_SHARED_FLOATS_PER_THREAD
#define CLOVER_SHARED_FLOATS_PER_THREAD 0
#endif

#ifndef NDEGTM_SHARED_FLOATS_PER_THREAD
#define NDEGTM_SHARED_FLOATS_PER_THREAD 0
#endif

  // these should not be namespaced!!
  // determines whether the temporal ghost zones are packed with a gather kernel,
  // as opposed to multiple calls to hipMemcpy()
  static bool kernelPackT = false;

  void setKernelPackT(bool packT) { kernelPackT = packT; }

  bool getKernelPackT() { return kernelPackT; }

  namespace dslash {
    int it = 0;

#ifdef PTHREADS
    hipEvent_t interiorDslashEnd;
#endif
    hipEvent_t packEnd[Nstream];
    hipEvent_t gatherStart[Nstream];
    hipEvent_t gatherEnd[Nstream];
    hipEvent_t scatterStart[Nstream];
    hipEvent_t scatterEnd[Nstream];
    hipEvent_t dslashStart;
    hipEvent_t dslashEnd;

    // FIX this is a hack from hell
    // Auxiliary work that can be done while waiting on comms to finis
    Worker *aux_worker;

#if CUDA_VERSION >= 8000
    cuuint32_t *commsEnd_h;
    hipDeviceptr_t commsEnd_d[Nstream];
#endif
  }

  void createDslashEvents()
  {
    using namespace dslash;
    // add hipEventDisableTiming for lower sync overhead
    for (int i=0; i<Nstream; i++) {
      hipEventCreate(&packEnd[i], hipEventDisableTiming);
      hipEventCreate(&gatherStart[i], hipEventDisableTiming);
      hipEventCreate(&gatherEnd[i], hipEventDisableTiming);
      hipEventCreateWithFlags(&scatterStart[i], hipEventDisableTiming);
      hipEventCreateWithFlags(&scatterEnd[i], hipEventDisableTiming);
    }
    hipEventCreateWithFlags(&dslashStart, hipEventDisableTiming);
    hipEventCreateWithFlags(&dslashEnd, hipEventDisableTiming);
#ifdef PTHREADS
    hipEventCreateWithFlags(&interiorDslashEnd, hipEventDisableTiming);
#endif

    aux_worker = NULL;

#if CUDA_VERSION >= 8000
    commsEnd_h = static_cast<cuuint32_t*>(mapped_malloc(Nstream*sizeof(int)));
    for (int i=0; i<Nstream; i++) {
      hipHostGetDevicePointer((void**)&commsEnd_d[i], commsEnd_h+i, 0);
      commsEnd_h[i] = 0;
    }
#endif

    checkCudaError();
  }


  void destroyDslashEvents()
  {
    using namespace dslash;

#if CUDA_VERSION >= 8000
    host_free(commsEnd_h);
    commsEnd_h = 0;
#endif

    for (int i=0; i<Nstream; i++) {
      hipEventDestroy(packEnd[i]);
      hipEventDestroy(gatherStart[i]);
      hipEventDestroy(gatherEnd[i]);
      hipEventDestroy(scatterStart[i]);
      hipEventDestroy(scatterEnd[i]);
    }

    hipEventDestroy(dslashStart);
    hipEventDestroy(dslashEnd);
#ifdef PTHREADS
    hipEventDestroy(interiorDslashEnd);
#endif

    checkCudaError();
  }

  using namespace dslash_aux;

#include <gamma5.h>		// g5 kernel
  
  /**
     Class for the gamma5 kernels, sFloat is the typename of the spinor components (double2, float4...)
  */

  template <typename sFloat>
  class Gamma5Cuda : public Tunable {
    
  private:
    cudaColorSpinorField *out;		//Output spinor
    const cudaColorSpinorField *in;		//Input spinor
    
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return in->X(0) * in->X(1) * in->X(2) * in->X(3); }
    
    char *saveOut, *saveOutNorm;
    
  public:
    Gamma5Cuda(cudaColorSpinorField *out, const cudaColorSpinorField *in) :
      out(out), in(in) {
      bindSpinorTex<sFloat>(in, out);
      dslashParam.out = out->V();
      dslashParam.outNorm = (float*)out->Norm();
      dslashParam.in = (void*)in->V();
      dslashParam.inNorm = (float*)in->Norm();
      dslashParam.sp_stride = in->Stride();
      strcpy(aux,"gamma5");
    }
    
    virtual ~Gamma5Cuda() { unbindSpinorTex<sFloat>(in, out); }
    
    TuneKey tuneKey() const
    {
      return TuneKey(in->VolString(), typeid(*this).name());
    }
    
    void apply(const hipStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (in->Precision() == QUDA_DOUBLE_PRECISION) {
	gamma5DKernel<<<tp.grid, tp.block, tp.shared_bytes>>> (dslashParam);
      } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
	gamma5SKernel<<<tp.grid, tp.block, tp.shared_bytes>>> (dslashParam);
      } else {
	errorQuda("Undefined for precision %d", in->Precision());
      }
    }
    
    void preTune()
    {
      saveOut = new char[out->Bytes()];
      hipMemcpy(saveOut, out->V(), out->Bytes(), hipMemcpyDeviceToHost);

      if (typeid(sFloat) == typeid(short4))
	{
	  saveOutNorm = new char[out->NormBytes()];
	  hipMemcpy(saveOutNorm, out->Norm(), out->NormBytes(), hipMemcpyDeviceToHost);
	}
    }
    
    void postTune()
    {
      hipMemcpy(out->V(), saveOut, out->Bytes(), hipMemcpyHostToDevice);
      delete[] saveOut;
      
      if (typeid(sFloat) == typeid(short4))
	{
	  hipMemcpy(out->Norm(), saveOutNorm, out->NormBytes(), hipMemcpyHostToDevice);
	  delete[] saveOutNorm;
	}
    }
    
    long long flops() const { return 12ll * in->VolumeCB(); }
    long long bytes() const { return in->Bytes() + in->NormBytes() + out->Bytes() + out->NormBytes(); }
  };

  /**
     Applies a gamma5 matrix to a spinor, this is the function to be called in interfaces and it requires only
     pointers to the output spinor (out) and the input spinor (in), in that order
  */

  void gamma5Cuda(cudaColorSpinorField *out, const cudaColorSpinorField *in)
  {
    dslashParam.threads = in->Volume();

    Tunable *gamma5 = 0;

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
      gamma5 = new Gamma5Cuda<double2>(out, in);
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      gamma5 = new Gamma5Cuda<float4>(out, in);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      errorQuda("Half precision not supported for gamma5 kernel yet");
    }

    gamma5->apply(streams[Nstream-1]);
    checkCudaError();

    delete gamma5;
  }

template <typename sFloat, typename cFloat>
class CloverCuda : public Tunable {
  private:
    cudaColorSpinorField *out;
    float *outNorm;
    char *saveOut, *saveOutNorm;
    const cFloat *clover;
    const float *cloverNorm;
    const cudaColorSpinorField *in;

  protected:
    unsigned int sharedBytesPerThread() const
    {
      int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
      return CLOVER_SHARED_FLOATS_PER_THREAD * reg_size;
    }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return in->VolumeCB(); }

  public:
    CloverCuda(cudaColorSpinorField *out, const cFloat *clover, const float *cloverNorm, 
	       int cl_stride, const cudaColorSpinorField *in)
      : out(out), clover(clover), cloverNorm(cloverNorm), in(in)
    {
      bindSpinorTex<sFloat>(in);

      dslashParam.out = (void*)out->V();
      dslashParam.outNorm = (float*)out->Norm();
      dslashParam.in = (void*)in->V();
      dslashParam.inNorm = (float*)in->Norm();
      dslashParam.clover = (void*)clover;
      dslashParam.cloverNorm = (float*)cloverNorm;

      dslashParam.sp_stride = in->Stride();
#ifdef GPU_CLOVER_DIRAC
      dslashParam.cl_stride = cl_stride;
#endif
    }
    virtual ~CloverCuda() { unbindSpinorTex<sFloat>(in); }
    void apply(const hipStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      dim3 gridDim( (dslashParam.threads+tp.block.x-1) / tp.block.x, 1, 1);
      if (typeid(sFloat)==typeid(double2)) {
	cloverDKernel <<<gridDim, tp.block, tp.shared_bytes, stream>>>(dslashParam);
      } else if (typeid(sFloat)==typeid(float4)) {
	cloverSKernel <<<gridDim, tp.block, tp.shared_bytes, stream>>>(dslashParam);
      } else {
	cloverHKernel <<<gridDim, tp.block, tp.shared_bytes, stream>>>(dslashParam);
      }
    }
    virtual TuneKey tuneKey() const { return TuneKey(in->VolString(), typeid(*this).name()); }

    // Need to save the out field if it aliases the in field
    void preTune() {
      if (in == out) {
        saveOut = new char[out->Bytes()];
        hipMemcpy(saveOut, out->V(), out->Bytes(), hipMemcpyDeviceToHost);
        if (typeid(sFloat) == typeid(short4)) {
          saveOutNorm = new char[out->NormBytes()];
          hipMemcpy(saveOutNorm, out->Norm(), out->NormBytes(), hipMemcpyDeviceToHost);
        }
      }
    }

    // Restore if the in and out fields alias
    void postTune() {
      if (in == out) {
        hipMemcpy(out->V(), saveOut, out->Bytes(), hipMemcpyHostToDevice);
        delete[] saveOut;
        if (typeid(sFloat) == typeid(short4)) {
          hipMemcpy(out->Norm(), saveOutNorm, out->NormBytes(), hipMemcpyHostToDevice);
          delete[] saveOutNorm;
        }
      }
    }

    long long flops() const { return 504ll * in->VolumeCB(); }
};


void cloverCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, const FullClover clover, 
		const cudaColorSpinorField *in, const int parity) {

  dslashParam.parity = parity;
  dslashParam.threads = in->Volume();

#ifdef GPU_CLOVER_DIRAC
  Tunable *clov = 0;
  void *cloverP, *cloverNormP;
  QudaPrecision clover_prec = bindCloverTex(clover, parity, &cloverP, &cloverNormP);

  if (in->Precision() != clover_prec)
    errorQuda("Mixing clover and spinor precision not supported");

  if (in->Precision() == QUDA_DOUBLE_PRECISION) {
    clov = new CloverCuda<double2, double2>(out, (double2*)cloverP, (float*)cloverNormP, clover.stride, in);
  } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
    clov = new CloverCuda<float4, float4>(out, (float4*)cloverP, (float*)cloverNormP, clover.stride, in);
  } else if (in->Precision() == QUDA_HALF_PRECISION) {
    clov = new CloverCuda<short4, short4>(out, (short4*)cloverP, (float*)cloverNormP, clover.stride, in);
  }
  clov->apply(0);

  unbindCloverTex(clover);
  checkCudaError();

  delete clov;
#else
  errorQuda("Clover dslash has not been built");
#endif
}


template <typename sFloat>
class TwistGamma5Cuda : public Tunable {

  private:
    cudaColorSpinorField *out;
    const cudaColorSpinorField *in;

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return in->X(0) * in->X(1) * in->X(2) * in->X(3); }

    char *saveOut, *saveOutNorm;

  public:
    TwistGamma5Cuda(cudaColorSpinorField *out, const cudaColorSpinorField *in,
        double kappa, double mu, double epsilon, const int dagger, QudaTwistGamma5Type twist) :
      out(out), in(in) 
  {
    bindSpinorTex<sFloat>(in);
    dslashParam.out = (void*)out->V();
    dslashParam.outNorm = (float*)out->Norm();
    dslashParam.in = (void*)in->V();
    dslashParam.inNorm = (float*)in->Norm();
    dslashParam.sp_stride = in->Stride();
    if(in->TwistFlavor() == QUDA_TWIST_SINGLET) {
      setTwistParam(dslashParam.a, dslashParam.b, kappa, mu, dagger, twist);
      dslashParam.c = 0.0;
#if (defined GPU_TWISTED_MASS_DIRAC) || (defined GPU_NDEG_TWISTED_MASS_DIRAC)
      dslashParam.fl_stride = in->VolumeCB();
#endif
    } else {//twist doublet
      dslashParam.a = kappa, dslashParam.b = mu, dslashParam.c = epsilon;
#if (defined GPU_TWISTED_MASS_DIRAC) || (defined GPU_NDEG_TWISTED_MASS_DIRAC)
      dslashParam.fl_stride = in->VolumeCB()/2;
#endif
    }
    dslashParam.a_f = dslashParam.a;
    dslashParam.b_f = dslashParam.b;
    dslashParam.c_f = dslashParam.c;
  }

    virtual ~TwistGamma5Cuda() { unbindSpinorTex<sFloat>(in); }

    TuneKey tuneKey() const { return TuneKey(in->VolString(), typeid(*this).name(), in->AuxString()); }

    void apply(const hipStream_t &stream) 
    {
#if (defined GPU_TWISTED_MASS_DIRAC) || (defined GPU_NDEG_TWISTED_MASS_DIRAC)
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      dim3 gridDim( (dslashParam.threads+tp.block.x-1) / tp.block.x, 1, 1);
      if(in->TwistFlavor() == QUDA_TWIST_SINGLET) {
        twistGamma5Kernel<sFloat,false><<<gridDim, tp.block, tp.shared_bytes, stream>>>(dslashParam);
      } else {
        twistGamma5Kernel<sFloat,true><<<gridDim, tp.block, tp.shared_bytes, stream>>>(dslashParam);
      }
#endif
    }

    void preTune() {
      saveOut = new char[out->Bytes()];
      hipMemcpy(saveOut, out->V(), out->Bytes(), hipMemcpyDeviceToHost);
      if (typeid(sFloat) == typeid(short4)) {
        saveOutNorm = new char[out->NormBytes()];
        hipMemcpy(saveOutNorm, out->Norm(), out->NormBytes(), hipMemcpyDeviceToHost);
      }
    }

    void postTune() {
      hipMemcpy(out->V(), saveOut, out->Bytes(), hipMemcpyHostToDevice);
      delete[] saveOut;
      if (typeid(sFloat) == typeid(short4)) {
        hipMemcpy(out->Norm(), saveOutNorm, out->NormBytes(), hipMemcpyHostToDevice);
        delete[] saveOutNorm;
      }
    }

    long long flops() const { return 24ll * in->VolumeCB(); }
    long long bytes() const { return in->Bytes() + in->NormBytes() + out->Bytes() + out->NormBytes(); }
};

//!ndeg tm: 
void twistGamma5Cuda(cudaColorSpinorField *out, const cudaColorSpinorField *in,
    const int dagger, const double &kappa, const double &mu, const double &epsilon,   const QudaTwistGamma5Type twist)
{
  if(in->TwistFlavor() == QUDA_TWIST_SINGLET)
    dslashParam.threads = in->Volume();
  else //twist doublet    
    dslashParam.threads = in->Volume() / 2;

#if (defined GPU_TWISTED_MASS_DIRAC) || (defined GPU_NDEG_TWISTED_MASS_DIRAC)
  Tunable *twistGamma5 = 0;

  if (in->Precision() == QUDA_DOUBLE_PRECISION) {
    twistGamma5 = new TwistGamma5Cuda<double2>(out, in, kappa, mu, epsilon, dagger, twist);
  } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
    twistGamma5 = new TwistGamma5Cuda<float4>(out, in, kappa, mu, epsilon, dagger, twist);
  } else if (in->Precision() == QUDA_HALF_PRECISION) {
    twistGamma5 = new TwistGamma5Cuda<short4>(out, in, kappa, mu, epsilon, dagger, twist);
  }

  twistGamma5->apply(streams[Nstream-1]);
  checkCudaError();

  delete twistGamma5;
#else
  errorQuda("Twisted mass dslash has not been built");
#endif // GPU_TWISTED_MASS_DIRAC
}

#if defined(GPU_TWISTED_CLOVER_DIRAC)
#include "dslash_core/tmc_gamma_core.h"
#endif

template <typename cFloat, typename sFloat>
class TwistCloverGamma5Cuda : public Tunable {
  private:
    const cFloat *clover;
    const float *cNorm;
    const cFloat *cloverInv;
    const float *cNrm2;
    QudaTwistGamma5Type twist;
    cudaColorSpinorField *out;
    const cudaColorSpinorField *in;

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return in->X(0) * in->X(1) * in->X(2) * in->X(3); }
    char *saveOut, *saveOutNorm;
    char aux_string[TuneKey::aux_n];

  public:
    TwistCloverGamma5Cuda(cudaColorSpinorField *out, const cudaColorSpinorField *in,
        double kappa, double mu, double epsilon, const int dagger, QudaTwistGamma5Type tw,
			  cFloat *clov, const float *cN, cFloat *clovInv, const float *cN2, int cl_stride) :
      clover(clov), cNorm(cN), cloverInv(clovInv), cNrm2(cN2), twist(tw), out(out), in(in)
    {
      bindSpinorTex<sFloat>(in);
      dslashParam.out = (void*)out->V();
      dslashParam.outNorm = (float*)out->Norm();
      dslashParam.in = (void*)in->V();
      dslashParam.inNorm = (float*)in->Norm();
      dslashParam.clover = (void*)clov;
      dslashParam.cloverNorm = (float*)cN;
      dslashParam.clover = (void*)clovInv;
      dslashParam.cloverNorm = (float*)cN2;
      dslashParam.sp_stride = in->Stride();
#ifdef GPU_TWISTED_CLOVER_DIRAC
      dslashParam.cl_stride = cl_stride;
      dslashParam.fl_stride = in->VolumeCB();
#endif

      if(in->TwistFlavor() == QUDA_TWIST_SINGLET) {
	setTwistParam(dslashParam.a, dslashParam.b, kappa, mu, dagger, tw);
      } else {//twist doublet
	errorQuda("ERROR: Non-degenerated twisted-mass not supported in this regularization\n");
      }
      dslashParam.a_f = dslashParam.a;
      dslashParam.b_f = dslashParam.b;

      strcpy(aux_string,in->AuxString());
      strcat(aux_string, twist == QUDA_TWIST_GAMMA5_DIRECT ? ",direct" : ",inverse");
    }

    virtual ~TwistCloverGamma5Cuda() { unbindSpinorTex<sFloat>(in); }

    TuneKey tuneKey() const {
      return TuneKey(in->VolString(), typeid(*this).name(), aux_string);
    }  

    void apply(const hipStream_t &stream)
    {
#if defined(GPU_TWISTED_CLOVER_DIRAC)
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      dim3 gridDim( (dslashParam.threads+tp.block.x-1) / tp.block.x, 1, 1);
      if(in->TwistFlavor() == QUDA_TWIST_SINGLET) {	//Idea for the kernel, two spinor inputs (IN and clover applied IN), on output (Clover applied IN + ig5IN)
        if (twist == QUDA_TWIST_GAMMA5_DIRECT)
          twistCloverGamma5Kernel<sFloat><<<gridDim, tp.block, tp.shared_bytes, stream>>>(dslashParam);
        else if (twist == QUDA_TWIST_GAMMA5_INVERSE)
          twistCloverGamma5InvKernel<sFloat><<<gridDim, tp.block, tp.shared_bytes, stream>>>(dslashParam);
      } else {
        errorQuda("ERROR: Non-degenerated twisted-mass not supported in this regularization\n");
      }
#endif
    }

    void preTune() {
      saveOut = new char[out->Bytes()];
      hipMemcpy(saveOut, out->V(), out->Bytes(), hipMemcpyDeviceToHost);
      if (typeid(sFloat) == typeid(short4)) {
        saveOutNorm = new char[out->NormBytes()];
        hipMemcpy(saveOutNorm, out->Norm(), out->NormBytes(), hipMemcpyDeviceToHost);
      }
    }

    void postTune() {
      hipMemcpy(out->V(), saveOut, out->Bytes(), hipMemcpyHostToDevice);
      delete[] saveOut;
      if (typeid(sFloat) == typeid(short4)) {
        hipMemcpy(out->Norm(), saveOutNorm, out->NormBytes(), hipMemcpyHostToDevice);
        delete[] saveOutNorm;
      }
    }


    long long flops() const { return 24ll * in->VolumeCB(); }	//TODO FIX THIS NUMBER!!!
    long long bytes() const { return in->Bytes() + in->NormBytes() + out->Bytes() + out->NormBytes(); }
};

void twistCloverGamma5Cuda(cudaColorSpinorField *out, const cudaColorSpinorField *in, const int dagger, const double &kappa, const double &mu,
    const double &epsilon, const QudaTwistGamma5Type twist, const FullClover *clov, const FullClover *clovInv, const int parity)
{
  if(in->TwistFlavor() == QUDA_TWIST_SINGLET)
    dslashParam.threads = in->Volume();
  else //twist doublet    
    errorQuda("Twisted doublet not supported in twisted clover dslash");

#ifdef GPU_TWISTED_CLOVER_DIRAC
  Tunable *tmClovGamma5 = 0;

  void *clover=0, *cNorm=0, *cloverInv=0, *cNorm2=0;
  QudaPrecision clover_prec = bindTwistedCloverTex(*clov, *clovInv, parity, &clover, &cNorm, &cloverInv, &cNorm2);

  if (in->Precision() != clover_prec)
    errorQuda("ERROR: Clover precision and spinor precision do not match\n");

#ifndef DYNAMIC_CLOVER
  if (clov->stride != clovInv->stride) 
    errorQuda("clover and cloverInv must have matching strides (%d != %d)", clov->stride, clovInv->stride);
#endif
    

  if (in->Precision() == QUDA_DOUBLE_PRECISION) {
    tmClovGamma5 = new TwistCloverGamma5Cuda<double2,double2>
      (out, in, kappa, mu, epsilon, dagger, twist, (double2 *) clover, (float *) cNorm, (double2 *) cloverInv, (float *) cNorm2, clov->stride);
  } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
    tmClovGamma5 = new TwistCloverGamma5Cuda<float4,float4>
      (out, in, kappa, mu, epsilon, dagger, twist, (float4 *) clover, (float *) cNorm, (float4 *) cloverInv, (float *) cNorm2, clov->stride);
  } else if (in->Precision() == QUDA_HALF_PRECISION) {
    tmClovGamma5 = new TwistCloverGamma5Cuda<short4,short4>
      (out, in, kappa, mu, epsilon, dagger, twist, (short4 *) clover, (float *) cNorm, (short4 *) cloverInv, (float *) cNorm2, clov->stride);
  }

  tmClovGamma5->apply(streams[Nstream-1]);
  checkCudaError();

  delete tmClovGamma5;
  unbindTwistedCloverTex(*clov);
#else
  errorQuda("Twisted clover dslash has not been built");
#endif // GPU_TWISTED_MASS_DIRAC
}

} // namespace quda

#include "contract.cu"
