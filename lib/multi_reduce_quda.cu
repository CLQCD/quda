#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <tune_quda.h>
#include <float_vector.h>
#include <color_spinor_field_order.h>
#include <int32_to_char.h>

//#define QUAD_SUM
#ifdef QUAD_SUM
#include <dbldbl.h>
#endif

#include <cub_helper.cuh>

template<typename> struct ScalarType { };
template<> struct ScalarType<double> { typedef double type; };
template<> struct ScalarType<double2> { typedef double type; };
template<> struct ScalarType<double3> { typedef double type; };

template<typename> struct Vec2Type { };
template<> struct Vec2Type<double> { typedef double2 type; };

#ifdef QUAD_SUM
#define QudaSumFloat doubledouble
#define QudaSumFloat2 doubledouble2
#define QudaSumFloat3 doubledouble3
template<> struct ScalarType<doubledouble> { typedef doubledouble type; };
template<> struct ScalarType<doubledouble2> { typedef doubledouble type; };
template<> struct ScalarType<doubledouble3> { typedef doubledouble type; };
template<> struct Vec2Type<doubledouble> { typedef doubledouble2 type; };
#else
#define QudaSumFloat double
#define QudaSumFloat2 double2
#define QudaSumFloat3 double3
#endif

// work around for Fermi
#if (__COMPUTE_CAPABILITY__ < 300)
#undef MAX_MULTI_BLAS_N
#define MAX_MULTI_BLAS_N 2
#endif

static void checkSpinor(const ColorSpinorField &a, const ColorSpinorField &b) {
  if (a.Length() != b.Length())
    errorQuda("lengths do not match: %lu %lu", a.Length(), b.Length());
  if (a.Stride() != b.Stride())
    errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());
}

static struct {
  const char *vol_str;
  const char *aux_str;
  char aux_tmp[quda::TuneKey::aux_n];
} blasStrings;

namespace quda {

  // hooks into tune.cpp variables for policy tuning
  typedef std::map<TuneKey, TuneParam> map;
  const map& getTuneCache();

  void disableProfileCount();
  void enableProfileCount();

  void setPolicyTuning(bool);

  namespace blas {

    hipStream_t* getStream();
    hipEvent_t* getReduceEvent();

    template <int writeX, int writeY, int writeZ, int writeW>
    struct write {
      static constexpr int X = writeX;
      static constexpr int Y = writeY;
      static constexpr int Z = writeZ;
      static constexpr int W = writeW;
    };

    namespace reduce {

      namespace multi {
#define BLAS_SPINOR // do not include ghost functions in Spinor class to reduce parameter space overhead
#include <texture.h>
      }

#include <multi_reduce_core.cuh>
#include <multi_reduce_core.h>

    } // namespace reduce

    /**
       Base class from which all reduction functors should derive.
    */
    template <int NXZ, typename ReduceType, typename Float2, typename FloatN>
    struct MultiReduceFunctor {

      //! pre-computation routine called before the "M-loop"
      virtual __device__ __host__ void pre() { ; }

      //! where the reduction is usually computed and any auxiliary operations
      virtual __device__ __host__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y,
							   FloatN &z, FloatN &w, const int i, const int j) = 0;

      //! post-computation routine called after the "M-loop"
      virtual __device__ __host__ void post(ReduceType &sum) { ; }

    };


    /**
       Return the real dot product of x and y
       Broken at the moment---need to update reDotProduct with permuting, etc of cDotProduct below.
    */
    template<typename ReduceType> __device__ __host__ void dot_(ReduceType &sum, const double2 &a, const double2 &b) {
      sum += (ReduceType)a.x*(ReduceType)b.x;
      sum += (ReduceType)a.y*(ReduceType)b.y;
    }

    template<typename ReduceType> __device__ __host__ void dot_(ReduceType &sum, const float2 &a, const float2 &b) {
      sum += (ReduceType)a.x*(ReduceType)b.x;
      sum += (ReduceType)a.y*(ReduceType)b.y;
    }

    template<typename ReduceType> __device__ __host__ void dot_(ReduceType &sum, const float4 &a, const float4 &b) {
      sum += (ReduceType)a.x*(ReduceType)b.x;
      sum += (ReduceType)a.y*(ReduceType)b.y;
      sum += (ReduceType)a.z*(ReduceType)b.z;
      sum += (ReduceType)a.w*(ReduceType)b.w;
    }

   template <int NXZ, typename ReduceType, typename Float2, typename FloatN>
    struct Dot : public MultiReduceFunctor<NXZ, ReduceType, Float2, FloatN> {
      typedef typename scalar<Float2>::type real;
      const int NYW;
      Dot(const reduce::coeff_array<Complex> &a, const reduce::coeff_array<Complex> &b, const reduce::coeff_array<Complex> &c, int NYW) : NYW(NYW) { ; }
      __device__ __host__ void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, const int i, const int j)
     { dot_<ReduceType>(sum,x,y); }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 2; } //! flops per element
    };

    void reDotProduct(double* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y){
#ifndef SSTEP
    errorQuda("S-step code not built\n");
#else
    switch(x.size()){
      case 1:
        reduce::multiReduceCuda<1,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 2:
        reduce::multiReduceCuda<2,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 3:
        reduce::multiReduceCuda<3,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 4:
        reduce::multiReduceCuda<4,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 5:
        reduce::multiReduceCuda<5,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 6:
        reduce::multiReduceCuda<6,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 7:
        reduce::multiReduceCuda<7,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 8:
        reduce::multiReduceCuda<8,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      /*case 9:
        reduce::multiReduceCuda<9,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 10:
        reduce::multiReduceCuda<10,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 11:
        reduce::multiReduceCuda<11,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 12:
        reduce::multiReduceCuda<12,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 13:
        reduce::multiReduceCuda<13,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 14:
        reduce::multiReduceCuda<14,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 15:
        reduce::multiReduceCuda<15,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 16:
        reduce::multiReduceCuda<16,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;*/
      default:
        errorQuda("Unsupported vector size");
        break;
    }
#endif // SSTEP
    // do a single multi-node reduction only once we have computed all local dot products
    const int Nreduce = x.size()*y.size();
    reduceDoubleArray((double*)result, Nreduce);
  }


    /**
       Returns complex-valued dot product of x and y
    */
    template<typename ReduceType>
    __device__ __host__ void cdot_(ReduceType &sum, const double2 &a, const double2 &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      sum.x += (scalar)a.x*(scalar)b.x;
      sum.x += (scalar)a.y*(scalar)b.y;
      sum.y += (scalar)a.x*(scalar)b.y;
      sum.y -= (scalar)a.y*(scalar)b.x;
    }

    template<typename ReduceType>
    __device__ __host__ void cdot_(ReduceType &sum, const float2 &a, const float2 &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      sum.x += (scalar)a.x*(scalar)b.x;
      sum.x += (scalar)a.y*(scalar)b.y;
      sum.y += (scalar)a.x*(scalar)b.y;
      sum.y -= (scalar)a.y*(scalar)b.x;
    }

    template<typename ReduceType>
    __device__ __host__ void cdot_(ReduceType &sum, const float4 &a, const float4 &b) {
      typedef typename ScalarType<ReduceType>::type scalar;
      sum.x += (scalar)a.x*(scalar)b.x;
      sum.x += (scalar)a.y*(scalar)b.y;
      sum.x += (scalar)a.z*(scalar)b.z;
      sum.x += (scalar)a.w*(scalar)b.w;
      sum.y += (scalar)a.x*(scalar)b.y;
      sum.y -= (scalar)a.y*(scalar)b.x;
      sum.y += (scalar)a.z*(scalar)b.w;
      sum.y -= (scalar)a.w*(scalar)b.z;
    }

    template <int NXZ, typename ReduceType, typename Float2, typename FloatN>
    struct Cdot : public MultiReduceFunctor<NXZ, ReduceType, Float2, FloatN> {
      typedef typename scalar<Float2>::type real;
      const int NYW;
      Cdot(const reduce::coeff_array<Complex> &a, const reduce::coeff_array<Complex> &b, const reduce::coeff_array<Complex> &c, int NYW) : NYW(NYW) { ; }
      __device__ __host__ inline void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, const int i, const int j)
      { cdot_<ReduceType>(sum,x,y); }
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };

    template <int NXZ, typename ReduceType, typename Float2, typename FloatN>
    struct CdotCopy : public MultiReduceFunctor<NXZ, ReduceType, Float2, FloatN> {
      typedef typename scalar<Float2>::type real;
      const int NYW;
      CdotCopy(const reduce::coeff_array<Complex> &a, const reduce::coeff_array<Complex> &b, const reduce::coeff_array<Complex> &c, int NYW) : NYW(NYW) { ; }
      __device__ __host__ inline void operator()(ReduceType &sum, FloatN &x, FloatN &y, FloatN &z, FloatN &w, const int i, const int j)
      { cdot_<ReduceType>(sum,x,y); if (i==j) w = y;}
      static int streams() { return 2; } //! total number of input and output streams
      static int flops() { return 4; } //! flops per element
    };

    // This function does the outer product of dot products... in column major.
    // There's a function below called 'cDotProduct' that flips it to row major.
    template <template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerDiagonal, typename writeDiagonal,
	      template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerOffDiagonal, typename writeOffDiagonal>
    void multiReduce_recurse(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y,
			     std::vector<ColorSpinorField*>&z, std::vector<ColorSpinorField*>&w, int i_idx, int j_idx, bool hermitian, unsigned int tile_size) {

      if (y.size() > tile_size) // if greater than max single-kernel size, split and recurse
      {
        // Do the recurse first.
        Complex* result0 = &result[0];
        Complex* result1 = &result[x.size()*(y.size()/2)];
        std::vector<ColorSpinorField*> y0(y.begin(), y.begin() + y.size()/2);
        std::vector<ColorSpinorField*> y1(y.begin() + y.size()/2, y.end());
        std::vector<ColorSpinorField*> w0(w.begin(), w.begin() + w.size()/2);
        std::vector<ColorSpinorField*> w1(w.begin() + w.size()/2, w.end());
        multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>(result0, x, y0, z, w0, i_idx, 2*j_idx+0, hermitian, tile_size);
        multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>(result1, x, y1, z, w1, i_idx, 2*j_idx+1, hermitian, tile_size);
      }
      else
      {
        double2* cdot = new double2[x.size()*y.size()];

	// if at bottom of recursion, return if on lower left
	if (x.size() <= tile_size && hermitian) {
	  if (j_idx < i_idx) { return; }
	}

        reduce::coeff_array<Complex> a, b, c;

	if (x.size() <= tile_size) {
        switch(x.size()){ // COMMENT HERE FOR COMPILE TIME
        case 1:
          reduce::multiReduceCuda<1,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#if MAX_MULTI_BLAS_N >= 2
        case 2:
          reduce::multiReduceCuda<2,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#if MAX_MULTI_BLAS_N >= 3
        case 3:
          reduce::multiReduceCuda<3,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#if MAX_MULTI_BLAS_N >= 4
        case 4:
          reduce::multiReduceCuda<4,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#if MAX_MULTI_BLAS_N >= 5
        case 5:
          reduce::multiReduceCuda<5,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#if MAX_MULTI_BLAS_N >= 6
        case 6:
          reduce::multiReduceCuda<6,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#if MAX_MULTI_BLAS_N >= 7
        case 7:
          reduce::multiReduceCuda<7,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#if MAX_MULTI_BLAS_N >= 8
        case 8:
          reduce::multiReduceCuda<8,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#if MAX_MULTI_BLAS_N >= 9
	case 9:
          reduce::multiReduceCuda<9,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#if MAX_MULTI_BLAS_N >= 10
        case 10:
          reduce::multiReduceCuda<10,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#if MAX_MULTI_BLAS_N >= 11
        case 11:
          reduce::multiReduceCuda<11,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#if MAX_MULTI_BLAS_N >= 12
        case 12:
          reduce::multiReduceCuda<12,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#if MAX_MULTI_BLAS_N >= 13
        case 13:
          reduce::multiReduceCuda<13,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#if MAX_MULTI_BLAS_N >= 14
        case 14:
          reduce::multiReduceCuda<14,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#if MAX_MULTI_BLAS_N >= 15
        case 15:
          reduce::multiReduceCuda<15,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#if MAX_MULTI_BLAS_N >= 16
        case 16:
          reduce::multiReduceCuda<16,double2,QudaSumFloat2,ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal,false>
	    (cdot, a, b, c, x, y, z, w, i_idx, j_idx );
          break;
#endif //16
#endif //15
#endif //14
#endif //13
#endif //12
#endif //11
#endif //10
#endif // 9
#endif // 8
#endif // 7
#endif // 6
#endif // 5
#endif // 4
#endif // 3
#endif // 2
	}
	} else {
          // split the problem and recurse. Splitting in x requires
          // memory reshuffling (unless y = 1).
          // Use a few temporary variables. 

          Complex* tmpmajor = new Complex[x.size()*y.size()];
          Complex* result0 = &tmpmajor[0];
          Complex* result1 = &tmpmajor[(x.size()/2)*y.size()];
          std::vector<ColorSpinorField*> x0(x.begin(), x.begin() + x.size()/2);
          std::vector<ColorSpinorField*> x1(x.begin() + x.size()/2, x.end());
          std::vector<ColorSpinorField*> z0(z.begin(), z.begin() + z.size()/2);
          std::vector<ColorSpinorField*> z1(z.begin() + z.size()/2, z.end());

          multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>(result0, x0, y, z0, w, 2*i_idx+0, j_idx, hermitian, tile_size);
          multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>(result1, x1, y, z1, w, 2*i_idx+1, j_idx, hermitian, tile_size);

          const unsigned int xlen0 = x.size()/2;
          const unsigned int xlen1 = x.size() - xlen0;
          const unsigned int ylen = y.size();

          // Copy back into result.
          int count = 0, count0 = 0, count1 = 0;
          for (unsigned int i = 0; i < ylen; i++)
          {
            for (unsigned int j = 0; j < xlen0; j++)
              result[count++] = result0[count0++];
            for (unsigned int j = 0; j < xlen1; j++)
              result[count++] = result1[count1++];
          }

          delete[] tmpmajor;
        }

	// we are at the leaf of the binary tree (e.g., we ran the kernel): perform the row-to-column-major transpose here.
        if (x.size() <= tile_size)
        {
          const unsigned int xlen = x.size();
          const unsigned int ylen = y.size();
          for (unsigned int j = 0; j < xlen; j++)
            for (unsigned int i = 0; i < ylen; i++)
              result[i*xlen+j] = Complex(cdot[j*ylen + i].x, cdot[j*ylen+i].y);
        }
        delete[] cdot;
      }
    }


    template <template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerDiagonal,
	      typename writeDiagonal,
	      template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerOffDiagonal,
	      typename writeOffDiagonal>
    class TileSizeTune : public Tunable {
      typedef std::vector<ColorSpinorField*> vec;
      Complex *result;
      vec &x, &y, &z, &w;
      bool hermitian;
      bool Anorm;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      unsigned int max_tile_size;

    public:
      TileSizeTune(Complex *result, vec &x, vec &y, vec &z, vec &w, bool hermitian, bool Anorm = false)
	: result(result), x(x), y(y), z(z), w(w), hermitian(hermitian), Anorm(Anorm), max_tile_size(1)
      {
      	strcpy(aux, "policy,");
      	strcat(aux, x[0]->AuxString());
      	strcat(aux, ",");
      	strcat(aux, y[0]->AuxString());
        if (hermitian) strcat(aux, ",hermitian");
        if (Anorm) strcat(aux, ",Anorm");
	strcat(aux,",n=");
	char size[8];
	i32toa(size, (int32_t)x.size());
	strcat(aux,size);
	strcat(aux,",m=");
	i32toa(size, (int32_t)y.size());
	strcat(aux,size);

      	// before we do policy tuning we must ensure the kernel
      	// constituents have been tuned since we can't do nested tuning
      	// FIXME this will break if the kernels are destructive - which they aren't here
	if (getTuning() && getTuneCache().find(tuneKey()) == getTuneCache().end()) {
	  disableProfileCount(); // purely for profiling reasons, don't want to profile tunings.

	  if ( x.size()==1 || y.size()==1 ) { // 1-d reduction

	    max_tile_size = std::min(MAX_MULTI_BLAS_N, (int)std::max(x.size(), y.size()));

	    // Make sure constituents are tuned.
	    for ( unsigned int tile_size=1; tile_size <= max_tile_size; tile_size++) {
	      multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>
		(result, x, y, z, w, 0, 0, hermitian, tile_size);
	    }

	  } else { // 2-d reduction

	    // max_tile_size should be set to the largest power of 2 less than
	    // MAX_MULTI_BLAS_N, since we have a requirement that the
	    // tile size is a power of 2.
	    unsigned int max_count = 0;
	    unsigned int tile_size_tmp = MAX_MULTI_BLAS_N;
	    while (tile_size_tmp != 1) { tile_size_tmp = tile_size_tmp >> 1; max_count++; }
	    tile_size_tmp = 1;
	    for (unsigned int i = 0; i < max_count; i++) { tile_size_tmp = tile_size_tmp << 1; }
	    max_tile_size = tile_size_tmp;

	    // Make sure constituents are tuned.
	    for ( unsigned int tile_size=1; tile_size <= max_tile_size && tile_size <= x.size() &&
		    (tile_size <= y.size() || y.size()==1) ; tile_size*=2) {
	      multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>
		(result, x, y, z, w, 0, 0, hermitian, tile_size);
	    }

	  }

      	  enableProfileCount();
      	  setPolicyTuning(true);
      	}
      }

      virtual ~TileSizeTune() { setPolicyTuning(false); }

      void apply(const hipStream_t &stream) {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        // tp.aux.x is where the tile size is stored. "tp" is the tuning struct.
        // it contains blocksize, grid size, etc. Since we're only tuning
        // a policy, we don't care about those sizes. That's why we only
        // tune "aux.x", which is the tile size. 
        multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>
          (result, x, y, z, w, 0, 0, hermitian, tp.aux.x);
      }

      // aux.x is the tile size
      bool advanceAux(TuneParam &param) const
      {

	if ( x.size()==1 || y.size()==1 ) { // 1-d reduction

	  param.aux.x++;
	  if ( (unsigned int)param.aux.x <= max_tile_size ) {
	    return true;
	  } else {
	    param.aux.x = 1;
	    return false;
	  }

	} else { // 2-d reduction

	  param.aux.x *= 2; // only tune powers of two (FIXME)

	  if ( (unsigned int)param.aux.x <= max_tile_size && param.aux.x <= (int)x.size() &&
	       param.aux.x <= (int)y.size() ) {
	    return true;
	  } else {
	    param.aux.x = 1; // reset to the beginning (which we'd need for multi-dimensional tuning)
	    return false;
	  }

	}
      }

      bool advanceTuneParam(TuneParam &param) const { return advanceAux(param); }

      void initTuneParam(TuneParam &param) const  {
      	Tunable::initTuneParam(param);
      	param.aux.x = 1; param.aux.y = 0; param.aux.z = 0; param.aux.w = 0;
      }

      void defaultTuneParam(TuneParam &param) const  {
      	Tunable::defaultTuneParam(param); // default is max tile size
        // max_tile_size is MAX_MULTI_BLAS_N rounded down to the nearest power of 2.
      	param.aux.x = max_tile_size; param.aux.y = 0; param.aux.z = 0; param.aux.w = 0;
      }

      TuneKey tuneKey() const {
        return TuneKey(x[0]->VolString(), typeid(*this).name(), aux);
      }

      long long flops() const { return 0; } // FIXME
      long long bytes() const { return 0; } // FIXME

      void preTune() { } // FIXME - use write to determine what needs to be saved
      void postTune() { } // FIXME - use write to determine what needs to be saved
    };

    void cDotProduct(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y){
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      // cDotProduct_recurse returns a column-major matrix.
      // To be consistent with the multi-blas functions, we should
      // switch this to row-major.
      TileSizeTune<Cdot,write<0,0,0,0>,Cdot,write<0,0,0,0> > tile(result_tmp, x, y, x, y, false);
      tile.apply(0);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce);

      // Switch from col-major to row-major
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = 0; i < ylen; i++)
          result[j*ylen+i] = result_tmp[i*xlen + j];

      delete[] result_tmp;
    }

    void hDotProduct(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y){
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (x.size() != y.size()) errorQuda("Cannot call Hermitian block dot product on non-square inputs");

      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      TileSizeTune<Cdot,write<0,0,0,0>,Cdot,write<0,0,0,0> > tile(result_tmp, x, y, x, y, true, false); // last false is b/c L2 norm
      tile.apply(0);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce); // FIXME - could optimize this for Hermiticity as well

      // Switch from col-major to row-major
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = j; i < ylen; i++) {
          result[j*ylen+i] = result_tmp[i*xlen + j];
          result[i*ylen+j] = conj(result_tmp[i*xlen + j]);
	}

      delete[] result_tmp;
    }

    // for (p, Ap) norms in CG which are Hermitian. 
    void hDotProduct_Anorm(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y){
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (x.size() != y.size()) errorQuda("Cannot call Hermitian block A-norm dot product on non-square inputs");

      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      TileSizeTune<Cdot,write<0,0,0,0>,Cdot,write<0,0,0,0> > tile(result_tmp, x, y, x, y, true, true); // last true is b/c A norm
      tile.apply(0);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce); // FIXME - could optimize this for Hermiticity as well

      // Switch from col-major to row-major
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = j; i < ylen; i++) {
          result[j*ylen+i] = result_tmp[i*xlen + j];
          result[i*ylen+j] = conj(result_tmp[i*xlen + j]);
  }

      delete[] result_tmp;
    }

    // takes the outer product of inner products between and y and copies y into z
    void cDotProductCopy(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y,
			 std::vector<ColorSpinorField*>&z){

#if 0
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (y.size() != z.size()) errorQuda("Cannot copy input y of size %lu into z of size %lu\n", y.size(), z.size());

      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      // When recursing, only the diagonal tiles will do the copy, the rest just do the outer product
      TileSizeTune<CdotCopy,write<0,0,0,1>,Cdot,write<0,0,0,0> > tile(result_tmp, x, y, x, y, true);
      tile.apply(0);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce);

      // Switch from col-major to row-major. 
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = 0; i < ylen; i++)
          result[j*ylen+i] = result_tmp[i*xlen + j];

      delete[] result_tmp;
#else
      errorQuda("cDotProductCopy not enabled");
#endif
    }

   } // namespace blas

} // namespace quda
