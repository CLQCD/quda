#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field_order.h>
#include <launch_kernel.cuh>
#include <cub_helper.cuh>

namespace quda {

using namespace gauge;

#ifdef GPU_GAUGE_TOOLS

  template <typename Mom>
  struct MomActionArg : public ReduceArg<double> {
    int threads; // number of active threads required
    Mom mom;
    int X[4]; // grid dimensions
    
    MomActionArg(const Mom &mom, const GaugeField &meta)
      : ReduceArg<double>(), mom(mom) {
      threads = meta.VolumeCB();
      for(int dir=0; dir<4; ++dir) X[dir] = meta.X()[dir];
    }
  };

  template<int blockSize, typename Float, typename Mom>
  __global__ void computeMomAction(MomActionArg<Mom> arg){
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int parity = threadIdx.y;
    double action = 0.0;
    
    if(x < arg.threads) {  
      // loop over direction
      for (int mu=0; mu<4; mu++) {
	Float v[10];
	arg.mom.load(v, x, mu, parity);

	double local_sum = 0.0;
	for (int j=0; j<6; j++) local_sum += v[j]*v[j];
	for (int j=6; j<9; j++) local_sum += 0.5*v[j]*v[j];
	local_sum -= 4.0;
	action += local_sum;
      }
    }
    
    // perform final inter-block reduction and write out result
    reduce2d<blockSize,2>(arg, action);
  }

  template<typename Float, typename Mom>
  class MomAction : TunableLocalParity {
    MomActionArg<Mom> &arg;
    const GaugeField &meta;

  private:
    unsigned int minThreads() const { return arg.threads; }

  public:
    MomAction(MomActionArg<Mom> &arg, const GaugeField &meta) : arg(arg), meta(meta) {}
    virtual ~MomAction () { }

    void apply(const hipStream_t &stream){
      if (meta.Location() == QUDA_CUDA_FIELD_LOCATION){
	arg.result_h[0] = 0.0;
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	LAUNCH_KERNEL_LOCAL_PARITY(computeMomAction, tp, stream, arg, Float, Mom);
      } else {
	errorQuda("CPU not supported yet\n");
      }
    }

    TuneKey tuneKey() const {
      std::stringstream aux;
      aux << "threads=" << arg.threads << ",prec="  << sizeof(Float);
      return TuneKey(meta.VolString(), typeid(*this).name(), aux.str().c_str());
    }

    long long flops() const { return 4*2*arg.threads*23; }
    long long bytes() const { return 4*2*arg.threads*arg.mom.Bytes(); }
  };

  template<typename Float, typename Mom>
  void momAction(const Mom mom, const GaugeField& meta, double &action) {
    MomActionArg<Mom> arg(mom, meta);
    MomAction<Float,Mom> momAction(arg, meta);

    momAction.apply(0);
    qudaDeviceSynchronize();

    comm_allreduce((double*)arg.result_h);
    action = arg.result_h[0];
  }
  
  template<typename Float>
  double momAction(const GaugeField& mom) {
    double action = 0.0;
    
    if (mom.Order() == QUDA_FLOAT2_GAUGE_ORDER) {
      if (mom.Reconstruct() == QUDA_RECONSTRUCT_10) {
	momAction<Float>(FloatNOrder<Float,10,2,10>(mom), mom, action);
      } else {
	errorQuda("Reconstruction type %d not supported", mom.Reconstruct());
      }
    } else {
      errorQuda("Gauge Field order %d not supported", mom.Order());
    }
    
    return action;
  }
#endif
  
  double computeMomAction(const GaugeField& mom) {
    double action = 0.0;
#ifdef GPU_GAUGE_TOOLS
    if (mom.Precision() == QUDA_DOUBLE_PRECISION) {
      action = momAction<double>(mom);
    } else if(mom.Precision() == QUDA_SINGLE_PRECISION) {
      action = momAction<float>(mom);
    } else {
      errorQuda("Precision %d not supported", mom.Precision());
    }
#else
    errorQuda("%s not build", __func__);
#endif
    return action;
  }


#ifdef GPU_GAUGE_TOOLS
  template<typename Float, typename Mom, typename Force>
  struct UpdateMomArg {
    int threads;
    Mom mom;
    Float coeff;
    Force force;
    int X[4]; // grid dimensions
    UpdateMomArg(Mom &mom, const Float &coeff, Force &force, GaugeField &meta)
      : threads(meta.VolumeCB()), mom(mom), coeff(coeff), force(force) {
      for (int dir=0; dir<4; ++dir) X[dir] = meta.X()[dir];
    }
  };

  template<typename Float, typename Mom, typename Force>
  __global__ void UpdateMomKernel(UpdateMomArg<Float, Mom, Force> arg) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int parity = threadIdx.y;
    Matrix<complex<Float>,3> m, f;
    while(x<arg.threads){
      for (int d=0; d<4; d++) {
	arg.mom.load(reinterpret_cast<Float*>(m.data), x, d, parity);
	arg.force.load(reinterpret_cast<Float*>(f.data), x, d, parity);

	m = m + arg.coeff * f;
	makeAntiHerm(m);

	arg.mom.save(reinterpret_cast<Float*>(m.data), x, d, parity); 
      }
      
      x += gridDim.x*blockDim.x;
    }
    return;
  } // UpdateMom

  
  template<typename Float, typename Mom, typename Force>
  class UpdateMom : TunableLocalParity {
    UpdateMomArg<Float, Mom, Force> &arg;
    const GaugeField &meta;

  private:
    unsigned int minThreads() const { return arg.threads; }

  public:
    UpdateMom(UpdateMomArg<Float,Mom,Force> &arg, const GaugeField &meta) : arg(arg), meta(meta) {}
    virtual ~UpdateMom () { }

    void apply(const hipStream_t &stream){
      if(meta.Location() == QUDA_CUDA_FIELD_LOCATION){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	UpdateMomKernel<Float,Mom,Force><<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
      } else {
	errorQuda("CPU not supported yet\n");
      }
    }

    TuneKey tuneKey() const {
      std::stringstream aux;
      aux << "threads=" << arg.threads << ",prec="  << sizeof(Float);
      return TuneKey(meta.VolString(), typeid(*this).name(), aux.str().c_str());
    }

    void preTune() { arg.mom.save();}
    void postTune() { arg.mom.load();}
    long long flops() const { return 4*2*arg.threads*(36+42); }
    long long bytes() const { return 4*2*arg.threads*(2*arg.mom.Bytes()+arg.force.Bytes()); }
  };

  template<typename Float, typename Mom, typename Force>
  void updateMomentum(Mom mom, Float coeff, Force force, GaugeField &meta) {
    UpdateMomArg<Float,Mom,Force> arg(mom, coeff, force, meta);
    UpdateMom<Float,Mom,Force> update(arg, meta);
    update.apply(0);
  }
  
  template <typename Float>
  void updateMomentum(GaugeField &mom, double coeff, GaugeField &force) {
    if (mom.Reconstruct() != QUDA_RECONSTRUCT_10)
      errorQuda("Momentum field with reconstruct %d not supported", mom.Reconstruct());

    if (force.Reconstruct() == QUDA_RECONSTRUCT_10) {
      updateMomentum<Float>(FloatNOrder<Float, 18, 2, 11>(mom), static_cast<Float>(coeff),
			      FloatNOrder<Float, 18, 2, 11>(force), force);
    } else if (force.Reconstruct() == QUDA_RECONSTRUCT_NO) {
      updateMomentum<Float>(FloatNOrder<Float, 18, 2, 11>(mom), static_cast<Float>(coeff),
			      FloatNOrder<Float, 18, 2, 18>(force), force);
    } else {
      errorQuda("Unsupported force reconstruction: %d", force.Reconstruct());
    }
    
  }
#endif // GPU_GAUGE_TOOLS

  void updateMomentum(GaugeField &mom, double coeff, GaugeField &force) {
#ifdef GPU_GAUGE_TOOLS
    if(mom.Order() != QUDA_FLOAT2_GAUGE_ORDER)
      errorQuda("Unsupported output ordering: %d\n", mom.Order());

    if (mom.Precision() != force.Precision()) 
      errorQuda("Mixed precision not supported: %d %d\n", mom.Precision(), force.Precision());

    if (mom.Precision() == QUDA_DOUBLE_PRECISION) {
      updateMomentum<double>(mom, coeff, force);
    } else {
      errorQuda("Unsupported precision: %d", mom.Precision());
    }      

    checkCudaError();
#else 
    errorQuda("%s not built", __func__);
#endif // GPU_GAUGE_TOOLS

    return;
  }


#ifdef GPU_GAUGE_TOOLS

  template<typename Float, typename Force, typename Gauge>
  struct ApplyUArg {
    int threads;
    Force force;
    Gauge U;
    int X[4]; // grid dimensions
    ApplyUArg(Force &force, Gauge &U, GaugeField &meta)
      : threads(meta.VolumeCB()), force(force), U(U) {
      for (int dir=0; dir<4; ++dir) X[dir] = meta.X()[dir];
    }
  };

  template<typename Float, typename Force, typename Gauge>
  __global__ void ApplyUKernel(ApplyUArg<Float,Force,Gauge> arg) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int parity = threadIdx.y;
    Matrix<complex<Float>,3> f, u;

    while (x<arg.threads) {
      for (int d=0; d<4; d++) {
	arg.force.load(reinterpret_cast<Float*>(f.data), x, d, parity);
	arg.U.load(reinterpret_cast<Float*>(u.data), x, d, parity);

	f = u * f;

	arg.force.save(reinterpret_cast<Float*>(f.data), x, d, parity);
      }

      x += gridDim.x*blockDim.x;
    }

    return;
  } // ApplyU


  template<typename Float, typename Force, typename Gauge>
  class ApplyU : TunableLocalParity {
    ApplyUArg<Float, Force, Gauge> &arg;
    const GaugeField &meta;

  private:
    unsigned int minThreads() const { return arg.threads; }

  public:
    ApplyU(ApplyUArg<Float,Force,Gauge> &arg, const GaugeField &meta) : arg(arg), meta(meta) {}
    virtual ~ApplyU () { }

    void apply(const hipStream_t &stream){
      if(meta.Location() == QUDA_CUDA_FIELD_LOCATION){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	ApplyUKernel<Float,Force,Gauge><<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
      } else {
	errorQuda("CPU not supported yet\n");
      }
    }

    TuneKey tuneKey() const {
      std::stringstream aux;
      aux << "threads=" << arg.threads << ",prec="  << sizeof(Float);
      return TuneKey(meta.VolString(), typeid(*this).name(), aux.str().c_str());
    }

    void preTune() { arg.force.save();}
    void postTune() { arg.force.load();}
    long long flops() const { return 4*2*arg.threads*198; }
    long long bytes() const { return 4*2*arg.threads*(2*arg.force.Bytes()+arg.U.Bytes()); }
  };

  template<typename Float, typename Force, typename Gauge>
  void applyU(Force force, Gauge U, GaugeField &meta) {
    ApplyUArg<Float,Force,Gauge> arg(force, U, meta);
    ApplyU<Float,Force,Gauge> applyU(arg, meta);
    applyU.apply(0);
    qudaDeviceSynchronize();
  }
  template <typename Float>
  void applyU(GaugeField &force, GaugeField &U) {
    if (force.Reconstruct() != QUDA_RECONSTRUCT_NO)
      errorQuda("Force field with reconstruct %d not supported", force.Reconstruct());

    if (U.Reconstruct() == QUDA_RECONSTRUCT_NO) {
      applyU<Float>(FloatNOrder<Float, 18, 2, 18>(force), FloatNOrder<Float, 18, 2, 18>(U), force);
    } else if (U.Reconstruct() == QUDA_RECONSTRUCT_NO) {
      applyU<Float>(FloatNOrder<Float, 18, 2, 18>(force), FloatNOrder<Float, 18, 2, 12>(U), force);
    } else {
      errorQuda("Unsupported gauge reconstruction: %d", U.Reconstruct());
    }

  }
#endif // GPU_GAUGE_TOOLS

  void applyU(GaugeField &force, GaugeField &U) {
#ifdef GPU_GAUGE_TOOLS
    if(force.Order() != QUDA_FLOAT2_GAUGE_ORDER)
      errorQuda("Unsupported output ordering: %d\n", force.Order());

    if (force.Precision() != U.Precision())
      errorQuda("Mixed precision not supported: %d %d\n", force.Precision(), U.Precision());

    if (force.Precision() == QUDA_DOUBLE_PRECISION) {
      applyU<double>(force, U);
    } else {
      errorQuda("Unsupported precision: %d", force.Precision());
    }

    checkCudaError();
#else
    errorQuda("%s not built", __func__);
#endif // GPU_GAUGE_TOOLS

    return;
  }

} // namespace quda
